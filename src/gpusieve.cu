#include "hip/hip_runtime.h"
/**********************************************************************
   This file file is part of GPUSIEVE
   Copyright (C) 2012 by Rocke Verser.  All rights reserved.
  
   GPUSIEVE is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 **********************************************************************/

/**********************************************************************
   A personal note from the orignal author:

   GPUSIEVE was originally written during January through March
   of 2012 by Rocke Verser.  To the best of my (Rocke Verser's)
   knowledge, there was no similar GPU software available to the
   public at the time of this release.  My original goals were to
   learn (always my goal) and to determine whether or not an
   efficient sieve could be implemented for GPU-style architectures.

   As such, I consider this release to be a *prototype*.  I assure
   you it contains defects, and I assure you it will require
   modifications to run on any platform different from my own.

   This software was written specifically with GIMPS (The Great
   Internet Mersenne Prime Search) in mind.  Sieving primes has
   been an interest of mine since I was a boy.  And finding
   factors of Mersenne numbers (which all have a very specific
   and special form) is a modern example of how sieving primes
   remains an important algorithm.  By the way, if Eratosthenes
   owned the patent on sieving, one would hope it has expired,
   by now.

   [The Sieve of Eratosthenes is a classic algorithm.  It was one
   of those algorithms that sparked my interest in mathematical
   computing, and which I actually first implemented as a boy to
   run on an IBM 1130 computer.  I did, in fact, succeed in
   printing a table of prime numbers up to 10 million on the
   IBM 1130 -- A computer with 8192 16-bit words of main memory.
   Using the same computer, I also managed to print the value of
   pi to 100,000 places after the decimal.  Some of the same tricks
   used to achieve those feats in the early 1970's still show
   through in modern mathematical algorithms, including what
   follows.  Very little is *really new*.  Things are just
   rewrapped and reinvented.]
 **********************************************************************/

#include <stdio.h>
#include <malloc.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "my_intrinsics.h"

// #define MAX_SIEVE_PRIMES      304 // Can't go smaller without code changes
// #define MAX_SIEVE_PRIMES      559
   #define MAX_SIEVE_PRIMES     1500 // Good starting value for tuning
// #define MAX_SIEVE_PRIMES   250000
// #define MAX_SIEVE_PRIMES 10000000 // Tested to this level, but slow

#define SIEVE_BITS (3*1024*1024)   // Size of the bitmap used for sieving
                                   // Must be a multiple of 32K

// Mersenne number should be an input.  But this is prototype code.
// #define MERSENNE_EXPONENT       2053  // Can't go smaller without code changes
// #define MERSENNE_EXPONENT    1000099
// #define MERSENNE_EXPONENT   56753239
// #define MERSENNE_EXPONENT   56755087
// #define MERSENNE_EXPONENT   56759389
   #define MERSENNE_EXPONENT   53785969
// #define MERSENNE_EXPONENT   15485959
// #define MERSENNE_EXPONENT  999999937
// #define MERSENNE_EXPONENT 0x7FFFFFFF  // Tested to this level

// Bit limits should be an input.  But this is prototype code.
#define QBITMIN 71            // Sieve for factors greater than 2^71
#define QBITMAX 72            // Sieve for factors less than 2^72

// #define or #undef the following to enable or disable running a subset of the classes
#define DEBUGSUBSET

#ifdef DEBUGSUBSET
#define MINKCLASS 4611        // Fewer classes permit faster debug runs
#define MAXKCLASS 4619        // Fewer classes permit faster debug runs
#else
#define MINKCLASS 0           // For production, must test all classes
#define MAXKCLASS (4620-1)    // For production, must test all classses
#endif


// Note:  At larger values of MAX_SIEVE_PRIMES, observed average number
// of candidates passing is slightly higher than theoretical shown.
// [This is when using atomic operations, with no missed sieve bits.]
// Not sure why.  When doing "normal" sieving, there's no point sieving
// for 11's at any level below 121.  I conjecture a similar effect may be
// happening here, but I haven't analyzed how it fits sieving for prime
// q=2*k*p+1.  (When sieving a 26-bit Mersenne number, for factors at
// the 72-bit level, at MAX_SIEVE_PRIMES=10000000, we tend to see about
// 14.3+% of the candidates passing.)

// Using non-atomic operations also results in values larger than shown,
// but for the entirely different reason that some sieved bits are
// simply lost.  If code is modified to use non-atomic operations,
// consideration must be given to how that will change these passing
// percentages, and to whether those changes will impact any decisions
// based on the "law of large numbers".

// at MAX_SIEVE_PRIMES =      304, 35.35% of SIEVE_BITS should pass
// at MAX_SIEVE_PRIMES =      559, 32.42% of SIEVE_BITS should pass
// at MAX_SIEVE_PRIMES =     1000, 30.04% of SIEVE_BITS should pass
// at MAX_SIEVE_PRIMES =     1500, 28.59% of SIEVE_BITS should pass
// at MAX_SIEVE_PRIMES =     2500, 26.96% of SIEVE_BITS should pass
// at MAX_SIEVE_PRIMES =    25000, 21.50% of SIEVE_BITS should pass
// at MAX_SIEVE_PRIMES =   250000, 17.93% of SIEVE_BITS should pass
// at MAX_SIEVE_PRIMES =  1000000, 16.32% of SIEVE_BITS should pass
// at MAX_SIEVE_PRIMES =  9000000, 14.30% of SIEVE_BITS should pass
// at MAX_SIEVE_PRIMES = 10000000, 14.22% of SIEVE_BITS should pass (works, but slow)

   #define MAX_TF_IN_SIEVE (SIEVE_BITS*37/100)    // Should be fine, based on the law of large numbers.
// #define MAX_TF_IN_SIEVE (SIEVE_BITS*31/100)    // Should be fine, at MAX_SIEVE_PRIMES >= 1500.
// #define MAX_TF_IN_SIEVE (SIEVE_BITS*100/100)   // Guaranteed safe.  Handles complete failure of sieving.
                                                  // If you decrease SIEVE_BITS, recalculate safety margin.

// #define MAX_STREAMS 1     // Works fine.  Least memory footprint.
   #define MAX_STREAMS 2     // Classic "double buffering".  Generally works well.
// #define MAX_STREAMS 3     // Good starting point for wringing out bugs.
// #define MAX_STREAMS 6
// #define MAX_STREAMS 10    // Upper limit is when you run out of memory
                             // Be cautious of video performance at high values

// #define or #undef the following to enable or disable major classes of debugging
#undef  DEBUGMALLOC        // Define this to verbosely show memory allocations
#undef  DEBUGTREE          // Define this to examine the tree
#undef  DEBUGBOUNDS        // Define this to verbosely examine the bounds on the sieve
#undef  DEBUGBDELTA        // Define this to fetch and examine the delta array
#undef  DEBUGBITMAP        // Define this to fetch and examine the bitmap
#undef  DEBUGCANDIDATES    // Define this to fetch and examine the list of candidates


#define AX_COLUMNS 1       // Number of entries in atomic index array
                           // Code only uses 1 atomic index at the moment.

typedef struct
{
  unsigned int d0, d1, d2;
} int96;

#include "cudahelper.h"         // Define CUDA "helper" functions

typedef struct                  // Context information for each stream's work
{
  hipStream_t stream;          // Space for cuda Stream Handles
  hipEvent_t linearized_event; // Event placed after linearization function
  unsigned int   exp;           // Mersenne exponent
  unsigned int   kclass;        // Class number
  unsigned int   csieve_bits;   // Number of bits in current sieve
           int96 ckstart;       // Lowest k-value for current sieve
           int96 cbstart;       // Lowest b-value for current sieve
           int96 cbend;         // Highest b-value for current sieve
} sievecontext;

sievecontext context[MAX_STREAMS];         // Space for one context per Stream Handle


// The only allocation of memory is at program start.  No other alocation
// or deallocation is required while this software runs.  We *are* so very
// sloppy!  If this prototype code is used where dynamic memory management
// is required, this needs to be cleaned up.
unsigned int *h_m1;
unsigned int *h_m2;
unsigned int *h_m3;
unsigned int *h_m4;
unsigned int *h_m5;
// igned int *h_m6;
unsigned int *h_m7;
unsigned int *h_m8;
unsigned int *h_m9;
unsigned int *h_m10;

// Following are created once, and remain constant for the life of the program
unsigned int *h_primep;         /* Host copy of primep array */
unsigned int *h_kncount;        /* Maximum number of hits within SIEVE_PRIMES for each prime */
unsigned int *h_ktree;          /* Tree of sum of kncount values */

// Device and host each have a copy of these constant structures
unsigned int *d_primep;         /* Device copy of primep array */
unsigned int *d_kncount;        /* Maximum number of hits within SIEVE_PRIMES for each prime */
unsigned int *d_ktree;          /* Tree of sum of kncount values */

// These structures are dynamic, with each stream owning its own structures
unsigned int *d_bdelta[MAX_STREAMS];    /* Offset to first multiple of a prime */
unsigned int *d_bitmapw[MAX_STREAMS];   /* Bitmap for sieving */
unsigned int *d_karray[MAX_STREAMS];    /* List of k-offsets to be trial-factored */
unsigned int *d_xaindexes[MAX_STREAMS]; /* atomic indices into karray */

// Following are copied back from device, for debugging purposes
unsigned int *h_bdelta[MAX_STREAMS];    /* Offset to first multiple of a prime */
unsigned int *h_bitmapw[MAX_STREAMS];   /* Bitmap for sieving */
unsigned int *h_karray[MAX_STREAMS];    /* List k-offsets to be trial-factored */
unsigned int *h_xaindexes[MAX_STREAMS]; /* atomic indices into karray */


__global__ void rcv_build_prime_tree(
        unsigned int *d_plist,  /* In: pointer to list of primes */
        unsigned int pcount,    /* number of elements in list */
        unsigned int kcount,    /* maximum number of bits in sieve table */
        unsigned int *d_kncount,/* Out: pointer to list of counts of k-values per prime */
        unsigned int *d_ktree   /* Out: pointer to tree of counts of k-values per prime */
        );

__global__ void rcv_init_class(
        unsigned int exp,       /* Mersenne exponent, p, of M(p) = 2^p-1 */
        unsigned int nclass,    // Number of classes  (must = 4620) */
        unsigned int kclass,    /* Number of this class (kstart mod nclasses) */
        int96        kstart,    /* Starting k-value.  Must=class (mod nclass) */
        unsigned int *d_plist,  /* In: Pointer to list of primes for sieving */
        unsigned int pcount,    /* In: Number of primes in sieve list */
        unsigned int *d_bdelta  /* Out: First bit, s.t. q is a multiple of the prime */
        );

__global__ void rcv_set_sieve_bits(
        unsigned int kcount,    /* number of bits in upcoming sieve */
        unsigned int *d_bitmapw /* bitmap for the sieve */
        );

__global__ void rcv_sieve_small_13_61(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta13, /* list of deltas, starting with 13 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        );

__global__ void rcv_sieve_small_67_127(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta67, /* list of deltas, starting with 67 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        );

__global__ void rcv_sieve_small_131_251(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta131, /* list of 23 deltas, starting with 131 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        );

__global__ void rcv_sieve_small_257_509(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta257,  /* list of 43 deltas for primes 257 through 509 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        );

__global__ void rcv_sieve_small_521_1021(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta521,  /* list of 75 deltas for primes 521 through 1021 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        );

__global__ void rcv_sieve_small_1031_2039(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta1031,  /* list of 137 deltas for primes 1031 through 2039 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        );

__global__ void rcv_sieve_primes(
        unsigned int tidoffseta,/* Offset from tid to first tid's element in tree */
        unsigned int tidoffsetz,/* Offset from tid to just past last tid's element in tree */
        unsigned int *d_plist,  /* pointer to list of primes */
        unsigned int pcount,    /* number of elements in list */
        int96        kstart,    /* lowest k-value in upcoming sieve */
        unsigned int kcount,    /* number of bits in upcoming sieve */
        unsigned int *d_bdelta, /* pointer to starting deltas per prime */
        unsigned int *d_kncount,/* pointer to count of k-values per prime */
        unsigned int *d_ktree,  /* pointer to tree of count of k-values per prime */
        unsigned int *d_bitmapw /* bitmap for the sieve */
        );

__global__ void rcv_reset_atomic_indexes(
        unsigned int width,          /* width of atomic index array */
        unsigned int *d_xaindexes    /* atomic index into array */
        );

__global__ void rcv_linearize_sieve(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bitmapw,/* bitmap for the sieve, 32-bit words */
        unsigned int *d_karray, /* linear array of k-values */
        unsigned int kasize,    /* number of spots in output array */
        unsigned int *d_kaindex /* atomic allocation index into karray */
        );


////////////////////////////////////////////////////////////////////////
//
// fillprimep -- This functions fills a linear array with small primes
//
////////////////////////////////////////////////////////////////////////

void fillprimep(unsigned int nump)
{
  unsigned int i;
  unsigned int j;
  unsigned int n;

  // Class mechanism implicitly deals with primes 2, 3, 5, 7, and 11
  // So, we skip the first five primes when filling our array.

  for (i=0, n=13; i<nump; n+=2)    /* Fill an array, starting at index i=0 */
  {
    j=0;
    if ((n%3 != 0) && (n%5 != 0) && (n%7 != 0) && (n%11 != 0))
        for (j=0; j<i; j+=1)       /* Check divisibility by previous primes */
    {
      if (n%h_primep[j] == 0)      /* If divisible, then n is not prime */
        break;
      if (h_primep[j]*h_primep[j] > n) /* We only have to test to sqrt(n) */
        j=i;                       /* Force exit from loop, with j>=i */
    }
    if (j>=i)                      /* If j<i, then not prime */
      h_primep[i++] = n;           /* If j>=i, then add new prime to array */
  }
  printf("h_primep[%u] = %u\n", i-1, h_primep[i-1]);  /* Print last prime */
}


////////////////////////////////////////////////////////////////////////
//
// InitApplication -- This function performs one-time initialization
//                    for this application.
//
////////////////////////////////////////////////////////////////////////

bool bPinGenericMemory;

void InitApplication()
{
  bool b;               // true if we are printing verbose debug info

  bPinGenericMemory = true;      // pin memory to allow async transfers

#ifdef DEBUGMALLOC
  b = true;
#else
  b = false;
#endif

  {
    int devID;
    devID = gpuDeviceInit(-1);
    printf("gpuDeviceInit returns deviceID:  %u\n",devID);
  }

  if (b) printf("Size of h_primep element:  %u\n", sizeof(*h_primep));
  if (b) printf("Size of malloc:  %u\n", sizeof(*h_primep)*MAX_SIEVE_PRIMES);

  AllocateHostMemory(bPinGenericMemory, &h_m1, &h_primep,  sizeof(*h_primep)*MAX_SIEVE_PRIMES);
  if (b) printf("h_primep      = %16.16lX\n", h_primep);

  AllocateHostMemory(bPinGenericMemory, &h_m3, &h_kncount, sizeof(*h_kncount)*MAX_SIEVE_PRIMES);
  if (b) printf("h_kncount     = %16.16lX\n", h_kncount);

  AllocateHostMemory(bPinGenericMemory, &h_m4, &h_ktree,   sizeof(*h_ktree)*(2*MAX_SIEVE_PRIMES));
  if (b) printf("h_ktree       = %16.16lX\n", h_ktree);

  for (int i=0; i<MAX_STREAMS; i+=1)
  {
    AllocateHostMemory(bPinGenericMemory, &h_m2, &h_bdelta[i], sizeof(*h_bdelta[i])*MAX_SIEVE_PRIMES);
    if (b) printf("h_bdelta[%d]   = %16.16lX\n", i, h_bdelta[i]);

    AllocateHostMemory(bPinGenericMemory, &h_m5, &h_bitmapw[i], sizeof(*h_bitmapw[i])*((SIEVE_BITS+31)/32));
    if (b) printf("h_bitmapw[%d]  = %16.16lX\n", i, h_bitmapw[i]);

    AllocateHostMemory(bPinGenericMemory, &h_m7, &h_karray[i],  sizeof(*h_karray[i] )*MAX_TF_IN_SIEVE);
    if (b) printf("h_karray[%d]   = %16.16lX\n", i, h_karray[i] );

    AllocateHostMemory(bPinGenericMemory, &h_m8, &h_xaindexes[i], sizeof(*h_xaindexes[i])*AX_COLUMNS);
    if (b) printf("h_xaindexes[%d]= %16.16lX\n", i, h_xaindexes[i] );
  }

  checkCudaErrors(hipMalloc(&d_primep, sizeof(*d_primep)*MAX_SIEVE_PRIMES));
  if (b) printf("d_primep      = %16.16lX\n", d_primep);

  checkCudaErrors(hipMalloc(&d_kncount, sizeof(*d_kncount)*MAX_SIEVE_PRIMES));
  if (b) printf("d_kncount     = %16.16lX\n", d_kncount);

  checkCudaErrors(hipMalloc(&d_ktree, sizeof(*d_ktree)*(2*MAX_SIEVE_PRIMES)));
  if (b) printf("d_ktree       = %16.16lx\n", d_ktree);

  for (int i=0; i<MAX_STREAMS; i+=1)
  {
    checkCudaErrors(hipMalloc(&d_bdelta[i], sizeof(*d_bdelta[i])*MAX_SIEVE_PRIMES));
    if (b) printf("d_bdelta[%d]   = %16.16lX\n", i, d_bdelta[i]);

    checkCudaErrors(hipMalloc(&d_bitmapw[i], 4*((SIEVE_BITS+31)/32)));
    if (b) printf("d_bitmapw[%d]  = %16.16lx\n", i, d_bitmapw[i]);

    checkCudaErrors(hipMalloc(&d_karray[i] , sizeof(*d_karray[i])*MAX_TF_IN_SIEVE));
    if (b) printf("d_karray[%d]   = %16.16lx\n", i, d_karray[i] );

    checkCudaErrors(hipMalloc(&d_xaindexes[i] , sizeof(*d_xaindexes[i])*AX_COLUMNS));
    if (b) printf("d_xaindexes[%d]= %16.16lx\n", i, d_xaindexes[i] );
  }

  // Compute a table of primes for sieving
  {
    fillprimep(MAX_SIEVE_PRIMES);     // once per program run.  These primes do not change.

    checkCudaErrors(hipMemcpyAsync(
            d_primep,
            h_primep,
            sizeof(*h_primep)*MAX_SIEVE_PRIMES,
            hipMemcpyHostToDevice));  // copy table to the device.
  }

  // We let a single thread block generate the tree
  int threadsPerBlock = 256;

  // printf("Calling rcv_build_prime_tree<<<%u,%u>>>\n",1, threadsPerBlock);
  rcv_build_prime_tree<<<1, threadsPerBlock>>> (
        d_primep,  /* in: Pointer to list of primes */
        MAX_SIEVE_PRIMES, /* number of elements in list */
        SIEVE_BITS, /* maximum number of bits in upcoming sieves */
        d_kncount, /* out: pointer to list of counts of k-values per prime */
        d_ktree);  /* out: pointer to tree of counts of k-values per prime */

  // Get a copy of the list of counts
  checkCudaErrors(hipMemcpyAsync(
          h_kncount,
          d_kncount,
          sizeof(*h_kncount)*MAX_SIEVE_PRIMES,
          hipMemcpyDeviceToHost));

  // Get a copy of the tree of counts
  checkCudaErrors(hipMemcpyAsync(
          h_ktree,
          d_ktree,
          sizeof(*h_kncount)*(2*MAX_SIEVE_PRIMES),
          hipMemcpyDeviceToHost));

  // Ensure the list of counts and the tree have been fetched
  checkCudaErrors(hipDeviceSynchronize());  // Gotta make sure the tree has been fetched
                                             // CUDA drivers spin.  But it's only once.  Sigh.

#ifdef DEBUGTREE
  {
    bool bShowListSamples = true;
    bool bShowTreeSamples = true;

    if (bShowListSamples)  // Show first and last element of list?
    {
      printf("h_primep[0] = %u, h_kncount[0] = %u\n", h_primep[0], h_kncount[0]);
      printf("h_primep[%u] = %u, h_kncount[%u] = %u\n", MAX_SIEVE_PRIMES-1, h_primep[MAX_SIEVE_PRIMES-1], MAX_SIEVE_PRIMES-1, h_kncount[MAX_SIEVE_PRIMES-1]);
    }

    if (bShowTreeSamples)  // Show samples from the tree?
    {
      for (int i=0; i<40; i+=1)                       // Show some elements near the root of the tree
        printf("h_ktree[%u] = %u\n", i, h_ktree[i]);

      for (int i=MAX_SIEVE_PRIMES; i<MAX_SIEVE_PRIMES+64; i+=1)  // Show some of the leaves of the tree
        printf("h_ktree[%u] = %u\n", i, h_ktree[i]);

      for (int i=MAX_SIEVE_PRIMES-1; i>0; i=i>>1)     // Navigate up from the last two leaves of the tree
        printf("h_ktree[%u] = h_ktree[%u] + h_ktree[%u] : %u = %u + %u\n",
            i, 2*i, 2*i+1, h_ktree[i], h_ktree[2*i], h_ktree[2*i+1]);
    }

    // Perform a very basic check that tree was built correctly
    for (int i=1; i<MAX_SIEVE_PRIMES; i+=1)
      if (h_ktree[i] != h_ktree[2*i] + h_ktree[2*i+1])
        printf("ERROR IN TREE:  [%u] != [%u]+[%u] : %u != %u + %u\n", i, 2*i, 2*i+1,
                h_ktree[i], h_ktree[2*i], h_ktree[2*i+1]);
  }
#endif

}


////////////////////////////////////////////////////////////////////////
//
// InitMersenne -- This function performs initialization once for
//                 each new Mersenne exponent we receive.
//
//    exp -- The prime Mersenne exponent.  Such as 1000099 or 323000323
//    qlowpower
//    qhighpower -- These specify the range of candidates sought.
//                    2^qlowpoer < q < 2^qhighpower
//
////////////////////////////////////////////////////////////////////////

int96 mul96by32(int96 x96, unsigned int y32)
{
  int96 t96;
  unsigned long long t0,t1,t2;
  t2 = (0llu+x96.d2)*y32;
  t96.d2 = t2&0x00000000ffffffffllu;
  t1 = (0llu+x96.d1)*y32;
  t96.d2 += t1>>32;
  t96.d1 = t1&0x00000000ffffffffllu;
  t0 = (0llu+x96.d0)*y32;
  t96.d1 += t0>>32;
  t96.d0 = t0&0x00000000ffffffffllu;
  if (t96.d1 < t0>>32)	// If sum is smaller than what we added, then we should carry
    t96.d2 += 1;
  return(t96);
}

int96 div96by32(int96 x96, unsigned int y32)
{
  // Does not handle divide-by-zero.
  int96 t96;
  unsigned long long t0,t1,t2;
  t2 = (0llu+x96.d2);     // 64-bit version of high-order word of dividend
  t96.d2 = t2/y32;        // Set h/o word of quotient.  Overflow not possible.
  t1 = (t2 % y32) << 32;  // Remainder becomes h/o part of next partial divide.
  t1 += (0llu+x96.d1);    // Add in middle word of dividend
  t96.d1 = t1/y32;        // Set middle word of quotient.  Overflow not possible.
  t0 = (t1 % y32) << 32;  // Remainder becomes h/o part of next partial divide.
  t0 += (0llu+x96.d0);    // Add in low-order word of dividend.
  t96.d0 = t0/y32;        // Set l/o word of quotient.  Overflow not possible.
                          // Remainder is discarded.
  return(t96);
}

unsigned int mod96by32(int96 x96, unsigned int y32)
{
  // Does not handle divide-by-zero.
  unsigned int t;
  unsigned long long t0,t1,t2;
  t2 = (0llu+x96.d2);     // 64-bit version of high-order word of dividend
  t1 = (t2 % y32) << 32;  // Remainder becomes h/o part of next partial divide.
  t1 += (0llu+x96.d1);    // Add in middle word of dividend
  t0 = (t1 % y32) << 32;  // Remainder becomes h/o part of next partial divide.
  t0 += (0llu+x96.d0);    // Add in low-order word of dividend.
  t  = (t0 % y32);        // Remainder is returned
  return(t);
}

int96 add96by32(int96 x96, unsigned int y32)
{
  int96 t96;
  t96 = x96;
  t96.d0 += y32;
  if (t96.d0 < y32)
  {
    t96.d1 += 1;
    if (t96.d1 < 1)
      t96.d2 += 1;
  }
  return(t96);
}

int96 add96by96(int96 x96, int96 y96)
{
  int96 t96;
  unsigned long long t;
  t = (0llu + x96.d0) + y96.d0;
  t96.d0 = t & 0x00000000ffffffff;
  t >>= 32;
  t += (0llu + x96.d1) + y96.d1;
  t96.d1 = t & 0x00000000ffffffff;
  t >>= 32;
  t += (0llu + x96.d2) + y96.d2;
  t96.d2 = t & 0x00000000ffffffff;
  return(t96);
}

int96 sub96by96(int96 x96, int96 y96)
{
  int96 t96;
  t96 = y96;
  t96.d0 ^= 0xffffffff;
  t96.d1 ^= 0xffffffff;
  t96.d2 ^= 0xffffffff;
  t96 = add96by32(t96, 1);
  t96 = add96by96(x96, t96);
  return(t96);
}

int96 sub96by32(int96 x96, unsigned int y32)
{
  int96 t96;
  t96 = x96;
  if (t96.d0 >= y32) {t96.d0 -= y32; return(t96);} // no borrow from l/o word
  t96.d0 -= y32;

  if (t96.d1 >= 1) {t96.d1 -= 1; return(t96);}     // no borrow from middle word
  t96.d1 -= 1;

  t96.d2 -= 1;
  return(t96);
}

int cmp96(int96 x96, int96 y96)
{
  int96 t96;
  int t;
  t96 = sub96by96(x96, y96);
  t = t96.d2;
  if (t == 0)
    t = t96.d1;
  if (t == 0)
    t = t96.d0;
  return (t);
}

char *cvt96hex24(int96 x96, char *s, int slength)
{
  snprintf(s, slength, "%8.8X%8.8X%8.8X", x96.d2, x96.d1, x96.d0);
  return (s);
}

int96 mqstart;        // 2^minbit
int96 mkstart;        // Lowest k, s.t. 2*k*p+1 >= mqstart

int96 mqend;          // 2^maxbit
int96 mkend;          // Highest k, s.t. 2*k*p+1 <= mqend

int96 mbstart;        // Lowest b, s.t.  4620*b >= mkstart
int96 mbend;          // Highest b, s.t.  4620*b <= mkend
unsigned int mbstartrem;  // 4620*mbstart + mbstartrem === mkstart.  (-4620 < mbstartrem <= 0)
unsigned int mbendrem;    // 4620*mbend + mbendrem === mkend.  (0 <= mbendrem < 4620)


void InitMersenne(unsigned int exp, unsigned int qminbits, unsigned int qmaxbits)
{

  unsigned int f;       // Factor of our Mersenne number
  bool bprime;
  bool b;               // true if we are printing verbose debug info

#ifdef DEBUGBOUNDS
  b = true;
#else
  b = false;
#endif

  // Rudimentary range check.
  // Can't handle exponents that are inherently sieved by classes.
  // Also can't handle exponents that overflow to 32 bits.
  if (exp < 13 || exp > 0x7fffffff)
  {
    printf("InitMersenne: exp=%u is outside supported range\n", exp);
    exit(1);
  }

  bprime = true;        // Assume caller has provided a prime exponent

  // Check for small divisors, which aren't in our list of primes for sieving
  if (bprime) {f=2; if (exp%f == 0) bprime=false;}
  if (bprime) {f=3; if (exp%f == 0) bprime=false;}
  if (bprime) {f=5; if (exp%f == 0) bprime=false;}
  if (bprime) {f=7; if (exp%f == 0) bprime=false;}
  if (bprime) {f=11; if (exp%f == 0) bprime=false;}

  // Check prime divisors, which we happen to have handy
  for (int i=0; bprime && i<MAX_SIEVE_PRIMES; i+=1)
    {
      f = h_primep[i];
      if (f*f > exp)  // if clear of all divisors to sqrt(exp), then it's prime
        break;
      if (exp%f == 0) bprime=false;
    }

  // Check higher odd divisors
  for (unsigned int j=h_primep[MAX_SIEVE_PRIMES-1]+2; bprime; j+=2)
    {
      f = j;
      if (f*f > exp)              // if clear of all divisors to sqrt(exp), then it's prime
        break;
      if (exp%f == 0) bprime=false;
    }

  if (!bprime)
  {
    printf("InitMersenne: exp=%u is not prime.  It is divisible by %u\n", exp, f);
    exit(1);
  }

  if (qminbits >= qmaxbits)
  {
    printf("InitMersenne: qminbits=%u >= qmaxbits=%u\n",
            qminbits, qmaxbits);
    exit(1);
  }

  if (qmaxbits > 95)
  {
    printf("InitMersenne: qmaxbits=%u, exceeds limit of 95 bits\n", qmaxbits);
    exit(1);
  }

  if (qminbits < 24)
  {
    printf("InitMersenne: qminbits=%u is below limit of 24 bits\n", qminbits);
    exit(1);
  }

  if (b) printf("    exp=%8.8X%8.8X%8.8X = %20llu\n", 0, 0, exp, exp);

  mqstart.d2 = 0;
  mqstart.d1 = 0;
  mqstart.d0 = 0;

  if (qminbits >= 64 && qminbits <= 95)
    mqstart.d2 = 1<<(qminbits-64);

  if (qminbits >= 32 && qminbits <= 63)
    mqstart.d1 = 1<<(qminbits-32);

  if (                  qminbits <= 31)
    mqstart.d0 = 1<<(qminbits   );

  if (b) if (mqstart.d2 == 0)
    printf("mqstart=%8.8X%8.8X%8.8X = %20llu\n", mqstart.d2, mqstart.d1, mqstart.d0, ((0llu+mqstart.d1)<<32)+mqstart.d0);
  else
    printf("mqstart=%8.8X%8.8X%8.8X\n", mqstart.d2, mqstart.d1, mqstart.d0);

  mqend.d2 = 0;
  mqend.d1 = 0;
  mqend.d0 = 0;

  if (qmaxbits >= 64 && qmaxbits <= 95)
    mqend.d2 = 1<<(qmaxbits-64);

  if (qmaxbits >= 32 && qmaxbits <= 63)
    mqend.d1 = 1<<(qmaxbits-32);

  if (                  qmaxbits <= 31)
    mqend.d0 = 1<<(qmaxbits   );

  if (b) if (mqend.d2 == 0)
    printf("mqend  =%8.8X%8.8X%8.8X = %20llu\n", mqend.d2, mqend.d1, mqend.d0, ((0llu+mqend.d1)<<32)+mqend.d0);
  else
    printf("mqend  =%8.8X%8.8X%8.8X\n", mqend.d2, mqend.d1, mqend.d0);


  // Compute lowest possible k, s.t. mqstart <= q=2*k*p+1
  {
    unsigned long long t;
    mkstart = div96by32(mqstart, 2);
    mkstart = div96by32(mkstart, exp);
    mkstart = add96by32(mkstart, 1);  // Safe to always round up because divisor (exp) is prime
    t = 0;
    if (mkstart.d2 == 0)
      t = ((0llu+mkstart.d1) << 32) + mkstart.d0;
    if (b) printf("mkstart=%8.8X%8.8X%8.8X = %20llu\n", mkstart.d2, mkstart.d1, mkstart.d0, t);
  }

  // Compute highest possible k, s.t. q=2*k*p+1 <= mqend
  {
    unsigned long long t;
    mkend = div96by32(mqend, 2);
    mkend = div96by32(mkend, exp);
    t = 0;
    if (mkend.d2 == 0)
      t = ((0llu+mkend.d1) << 32) + mkend.d0;
    if (b) printf("mkend  =%8.8X%8.8X%8.8X = %20llu\n", mkend.d2, mkend.d1, mkend.d0, t);
  }


  // Compute lowest possible b, s.t. mkstart <= 4960*b
  {
    unsigned long long t2;
    mbstart = div96by32(mkstart, 4620);
    mbstartrem = mkstart.d0 - mbstart.d0*4620;     // Result guaranteed to be 0 <= t1 < 4620
    t2 = 0;
    if (mbstart.d2 == 0)
      t2 = ((0llu+mbstart.d1)<< 32) + mbstart.d0;
    if (b) printf("mbstart=%8.8X%8.8X%8.8X = %20llu | %4d\n", mbstart.d2, mbstart.d1, mbstart.d0, t2, mbstartrem);
  }

  // Compute highest possible b, s.t. 4960*b <= mkend
  {
    unsigned long long t2;
    mbend = div96by32(mkend, 4620);
    mbendrem = mkend.d0 - mbend.d0*4620;     // Result guaranteed to be 0 <= t1 < 4620
    t2 = 0;
    if (mbend.d2 == 0)
      t2 = ((0llu+mbend.d1)<< 32) + mbend.d0;
    if (b) printf("mbend  =%8.8X%8.8X%8.8X = %20llu | %4llu\n", mbend.d2, mbend.d1, mbend.d0, t2, mbendrem);
  }

  // Perform a quick test that our computed values are sensible
  {
    int96 tb1,                tb5;
    int96 tk1, tk2,      tk4, tk5;
    int96 tq1, tq2, tq3, tq4, tq5;

    char s1[25];
    char s2[25];
    char s3[25];

    // When scaled to corresponding values, we want the following relationships:
    //   bstart-1 <  kstart-1 <  qstart <= kstart   <= bstart
    //   bend     <= kend     <= qend   <  kend+1   <  bend+1

    // First, we handle the low end of the range, near minbits
    //   bstart-1 <  kstart-1 <  qstart <= kstart   <= bstart

    // In terms of b, please
    tb1 = mbstart;           // Compute 1 less than starting value
    tb1 = sub96by32(tb1, 1);
    tb5 = mbstart;           // Grab a copy of starting value

    // In terms of k, please
    tk1 = mul96by32(tb1, 4620);
    tk1 = add96by32(tk1, mbstartrem);
    tk2 = mkstart;           // Compute 1 less than starting value
    tk2 = sub96by32(tk2, 1);
    tk4 = mkstart;           // Grab a copy of starting value
    tk5 = mul96by32(tb5, 4620);
    tk5 = add96by32(tk5, mbstartrem);

    // In terms of q, please
    tq1 = mul96by32(tk1, exp);
    tq1 = mul96by32(tq1, 2);
    tq1 = add96by32(tq1, 1);
    tq2 = mul96by32(tk2, exp);
    tq2 = mul96by32(tq2, 2);
    tq2 = add96by32(tq2, 1);
    tq3 = mqstart;
    tq4 = mul96by32(tk4, exp);
    tq4 = mul96by32(tq4, 2);
    tq4 = add96by32(tq4, 1);
    tq5 = mul96by32(tk5, exp);
    tq5 = mul96by32(tq5, 2);
    tq5 = add96by32(tq5, 1);

    if (b) printf("     q1=%s  %s  %s  R %4u  = mbstart-1\n", cvt96hex24(tq1, s1, 25), cvt96hex24(tk1, s2, 25), cvt96hex24(tb1, s3, 25)+8, mbstartrem);
    if (b) printf("     q2=%s  %s  %24s  = mkstart-1\n", cvt96hex24(tq2, s1, 25), cvt96hex24(tk2, s2, 25), "");
    if (b) printf("     q3=%s  %24s  %24s  = mqstart\n",     cvt96hex24(tq3, s1, 25), "", "");
    if (b) printf("     q4=%s  %s  %24s  = mkstart\n", cvt96hex24(tq4, s1, 25), cvt96hex24(tk4, s2, 25), "");
    if (b) printf("     q5=%s  %s  %s  R %4u  = mbstart\n", cvt96hex24(tq5, s1, 25), cvt96hex24(tk5, s2, 25), cvt96hex24(tb5, s3, 25)+8, mbstartrem);

    if (cmp96(tq2,tq3) > 0 || cmp96(tq2,tq3) > 0)
    {
      printf("Computed mkstart is out of bounds.\n");
      exit(1);
    }

    if (cmp96(tq1,tq2) > 0 || cmp96(tq4,tq5) != 0)
    {
      printf("Computed mbstart is out of bounds.\n");
      exit(1);
    }


    // Next, we handle the high end of the range, near qmaxbits
    //   bend     <= kend     <= qend   <  kend+1   <  bend+1

    // In terms of b, please
    tb1 = mbend;             // Grab a copy of starting value
    tb5 = mbend;             // Compute 1 more than starting value
    tb5 = add96by32(tb5, 1);

    // In terms of k, please
    tk1 = mul96by32(tb1, 4620);
    tk1 = add96by32(tk1, mbendrem);
    tk2 = mkend;             // Grab a copy of starting value
    tk4 = mkend;             // Compute 1 more than starting value
    tk4 = add96by32(tk4, 1);
    tk5 = mul96by32(tb5, 4620);
    tk5 = add96by32(tk5, mbendrem);

    // In terms of q, please
    tq1 = mul96by32(tk1, exp);
    tq1 = mul96by32(tq1, 2);
    tq1 = add96by32(tq1, 1);
    tq2 = mul96by32(tk2, exp);
    tq2 = mul96by32(tq2, 2);
    tq2 = add96by32(tq2, 1);
    tq3 = mqend;  
    tq4 = mul96by32(tk4, exp);
    tq4 = mul96by32(tq4, 2);
    tq4 = add96by32(tq4, 1);
    tq5 = mul96by32(tk5, exp);
    tq5 = mul96by32(tq5, 2);
    tq5 = add96by32(tq5, 1);

    if (b) printf("\n");
    if (b) printf("     q1=%s  %s  %s  R %4u  = mbend\n", cvt96hex24(tq1, s1, 25), cvt96hex24(tk1, s2, 25), cvt96hex24(tb1, s3, 25)+8, mbendrem);
    if (b) printf("     q2=%s  %s  %24s  = mkend\n", cvt96hex24(tq2, s1, 25), cvt96hex24(tk2, s2, 25), "");
    if (b) printf("     q3=%s  %24s  %24s  = mqend\n",     cvt96hex24(tq3, s1, 25), "", "");
    if (b) printf("     q4=%s  %s  %24s  = mkend+1\n", cvt96hex24(tq4, s1, 25), cvt96hex24(tk4, s2, 25), "");
    if (b) printf("     q5=%s  %s  %s  R %4u  = mbend+1\n", cvt96hex24(tq5, s1, 25), cvt96hex24(tk5, s2, 25), cvt96hex24(tb5, s3, 25)+8, mbendrem);

    if (cmp96(tq2,tq3) > 0 || cmp96(tq2,tq3) > 0)
    {
      printf("Computed mkend is out of bounds.\n");
      exit(1);
    }

    if (cmp96(tq1,tq2) != 0 || cmp96(tq4,tq5) > 0)
    {
      printf("Computed mbend is out of bounds.\n");
      exit(1);
    }
  }
}

////////////////////////////////////////////////////////////////////////
//
// TestClass -- tests whether or not this class is composite
//
//    exp -- The prime Mersenne exponent.  Such as 1000099 or 56753239
//    class -- The proposed class (0 <= class < 4620)
//
////////////////////////////////////////////////////////////////////////

bool TestClass(unsigned int exp, unsigned int kclass)
{
  // q = 2*k*p+1
  unsigned int qclass;

  qclass = (((2 * kclass) * (exp % 9240)) + 1) % 9240;

  if ((qclass%8 != 1) && (qclass%8 != 7))
    return (false);     // qclass mod 8 not in {1,7}

  if (qclass%3 == 0)
    return (false);     // qclass mod 3 == 0

  if (qclass%5 == 0)
    return (false);     // qclass mod 5 == 0

  if (qclass%7 == 0)
    return (false);     // qclass mod 7 == 0

  if (qclass%11 == 0)
    return (false);     // qclass mod 11 == 0

  printf("TestClass: kclass=%4u --> qclass=%4u, qmod3=%u, qmod5=%u, qmod7=%u, qmod11=%2u, qmod8=%u\n",
            kclass, qclass, qclass%3, qclass%5, qclass%7, qclass%11, qclass%8);
  return (true);
}

////////////////////////////////////////////////////////////////////////
//
// InitClass -- This function performs initialization for a new class
//
//    exp -- The prime Mersenne exponent.  Such as 1000099 or 56753239
//    class -- The upcoming class (0 <= class < 4620)
//
////////////////////////////////////////////////////////////////////////

//    cqstart;        // 2^minbit
int96 ckstart;        // Lowest k, s.t. 2*k*p+1 >= qstart
int96 cbstart;        // Lowest b, s.t.  4620*b >= kstart

//    cqend;          // 2^maxbit
int96 ckend;          // Highest k, s.t. 2*k*p+1 <= qend
int96 cbend;          // Highest b, s.t.  4620*b <= kend

unsigned int csieve_bits;      // Total number of sieve bits to be processed for this class

void InitClass(unsigned int exp, unsigned int kclass)
{
  cbstart = mbstart;  // Our class bit-offset will start near the others
  if (kclass < mbstartrem)
    cbstart = add96by32(cbstart, 1);   // The low class numbers start one bit later

  ckstart = mul96by32(cbstart, 4620);
  ckstart = add96by32(ckstart, kclass);

  cbend = mbend;      // Our class bit-offset will end near the others
  if (kclass > mbendrem)
    cbend = sub96by32(cbend, 1);     // The high class numbers end one bit earlier

  int96 tq1, tq2, tq3, tq4, tq5;
  int96 tk1, tk2,      tk4, tk5;
  int96 tb1,                tb5;

  {
    tb1 = sub96by32(cbstart, 1);
    tb5 = cbstart;

    tk1 = mul96by32(tb1, 4620);
    tk1 = add96by32(tk1, kclass);
    tk4 = mkstart;
    tk5 = mul96by32(tb5, 4620);
    tk5 = add96by32(tk5, kclass);

    tq1 = mul96by32(tk1, exp);
    tq1 = mul96by32(tq1, 2);
    tq1 = add96by32(tq1, 1);
    tq3 = mqstart; 
    tq4 = mul96by32(tk4, exp);
    tq4 = mul96by32(tq4, 2);
    tq4 = add96by32(tq4, 1);
    tq5 = mul96by32(tk5, exp);
    tq5 = mul96by32(tq5, 2);
    tq5 = add96by32(tq5, 1);
#if 0
    {
      char s1[25], s2[25], s3[25];
      printf("\n");
      printf("     q1=%s  %s  %s  R %4u  = cbstart-1\n", cvt96hex24(tq1, s1, 25), cvt96hex24(tk1, s2, 25), cvt96hex24(tb1, s3, 25)+8, kclass);
      printf("     q3=%s  %24s  %24s  = mqstart\n",     cvt96hex24(tq3, s1, 25), "", "");
      printf("     q4=%s  %s  %24s  = mkstart\n", cvt96hex24(tq4, s1, 25), cvt96hex24(tk4, s2, 25), "");
      printf("     q5=%s  %s  %s  R %4u  = cbstart\n", cvt96hex24(tq5, s1, 25), cvt96hex24(tk5, s2, 25), cvt96hex24(tb5, s3, 25)+8, kclass);
    }
#endif

    if (cmp96(tq1,tq3) >= 0 || cmp96(tq4,tq5) > 0)
    {
      printf("Computed cbstart is out of bounds.\n");
      exit(1);
    }


    tb1 = cbend;
    tb5 = add96by32(cbend, 1);

    tk1 = mul96by32(tb1, 4620);
    tk1 = add96by32(tk1, kclass);
    tk2 = mkend;
    tk5 = mul96by32(tb5, 4620);
    tk5 = add96by32(tk5, kclass);

    tq1 = mul96by32(tk1, exp);
    tq1 = mul96by32(tq1, 2);
    tq1 = add96by32(tq1, 1);
    tq2 = mul96by32(tk2, exp);
    tq2 = mul96by32(tq2, 2);
    tq2 = add96by32(tq2, 1);
    tq3 = mqend; 
    tq5 = mul96by32(tk5, exp);
    tq5 = mul96by32(tq5, 2);
    tq5 = add96by32(tq5, 1);
#if 0
    {
      char s1[25], s2[25], s3[25];
      printf("\n");
      printf("     q1=%s  %s  %s  R %4u  = cbend\n", cvt96hex24(tq1, s1, 25), cvt96hex24(tk1, s2, 25), cvt96hex24(tb1, s3, 25)+8, kclass);
      printf("     q2=%s  %s  %24s  = mkend\n", cvt96hex24(tq2, s1, 25), cvt96hex24(tk2, s2, 25), "");
      printf("     q3=%s  %24s  %24s  = mqend\n",     cvt96hex24(tq3, s1, 25), "", "");
      printf("     q5=%s  %s  %s  R %4u  = cbend+1\n", cvt96hex24(tq5, s1, 25), cvt96hex24(tk5, s2, 25), cvt96hex24(tb5, s3, 25)+8, kclass);
    }
#endif

    if (cmp96(tq1,tq2) > 0 || cmp96(tq3,tq5) >= 0)
    {
      printf("Computed cbend is out of bounds.\n");
      exit(1);
    }
  }
}


////////////////////////////////////////////////////////////////////////
//
// PrepClassBatch -- This function prepares for a batch of sieving for
//                   the class last InitClass'ed
//
//    exp -- The prime Mersenne exponent.  Such as 1000099 or 56753239
//    class -- The upcoming class (0 <= class < 4620)
//
////////////////////////////////////////////////////////////////////////

void PrepClassBatch(unsigned int exp, unsigned int kclass, int streamix)
{
  // We need to cooperate with the "sieve_small" and "linearize"
  // kernels on the trailing blocks.  Those kernels expect a minimum
  // power-of-2 candidates per block.  We must always give them a
  // multiple of their candidates per block.

  const unsigned int LCPB = 32*1024;  // Largest Candidates per Block
                                      // Several of the rcv_sieve_small_xxx kernels
                                      // are configured for 32K candidates per block.

  int96 t96;
  t96 = sub96by96(cbend, cbstart);
  t96 = add96by32(t96, 1);

  if (t96.d2 != 0 || t96.d1 != 0 || (t96.d0/2) >= SIEVE_BITS)
    csieve_bits = SIEVE_BITS;    // With plenty of bits to go, process the maximum possible
  else                           // We even up the last 2 sieves of this class, hoping for better latency
    if (t96.d0 > SIEVE_BITS)     // Can we finish up on this sieve?
      csieve_bits = ((t96.d0+1)/2+LCPB-1) & ~(LCPB-1);// No.  Do half, rounded up to multiple of 2^k bits.
    else
      csieve_bits = (t96.d0+LCPB-1) & ~(LCPB-1);  // Get 'er done, rounding up to multiple of 2^k bits.
                                 // ***Caution*** We may exceed qmaxbits.  We may
                                 // test some candidates beyond caller-specified range.
                                 // Consider shifting the last block down so it ends at qmaxbits.  A
                                 // very little bit of retesting, rather than overrunning qmaxbits.

  int threadsPerBlock = 256;
  int blocksPerGrid;
  blocksPerGrid = (MAX_SIEVE_PRIMES + threadsPerBlock - 1) / threadsPerBlock;

  // printf("Calling rcv_init_class<<<%u,%u>>>\n", blocksPerGrid, threadsPerBlock);
  rcv_init_class <<<blocksPerGrid, threadsPerBlock, 0, context[streamix].stream>>> (
      exp,                      /* p of M(p)=2^p-1 */
      4620,                     /* Number of classes.  4620 = 3 * 5 * 7 * 11 * 4 */
      kclass,                   /* Class number of this run */
      ckstart,                  /* First 96-bit k-value of this run.  Must be congruent to class number. */
      d_primep,                 /* In: Pointer to list of prime numbers for sieving */
      MAX_SIEVE_PRIMES,         /* Number of primes in the sieve-list */
      d_bdelta[streamix]);      /* Out: per-prime.  Returns first bit, s.t. q is a multiple of the prime */

  // Since we are running multiple (possibly overlapped) streams, we remember
  // the important context information about what our stream is running.
  context[streamix].exp         = exp;
  context[streamix].kclass      = kclass;
  context[streamix].ckstart     = ckstart;
  context[streamix].cbstart     = cbstart;
  context[streamix].cbend       = cbend;
  context[streamix].csieve_bits = csieve_bits;
}


////////////////////////////////////////////////////////////////////////
//
// NextClassBatch -- This function prepares for an additional batch
//                   of sieving for the class last InitClass'ed
//
//    exp -- The prime Mersenne exponent.  Such as 1000099 or 56753239
//    class -- The upcoming class (0 <= class < 4620)
//
//    Returns true if more work is required to complete this class.
//    Returns false if the class is complete.
//
////////////////////////////////////////////////////////////////////////

bool NextClassBatch(unsigned int exp, unsigned int kclass)
{

  cbstart = add96by32(cbstart, csieve_bits);  // Next sieve should begin this many bits ahead
  if (cmp96(cbstart, cbend) > 0)    // Is our work done?
    return (false);                 // Yes, return to caller

  // Recompute ckstart in preparation for next batch
  ckstart = mul96by32(cbstart, 4620);
  ckstart = add96by32(ckstart, kclass);

  return (true);                    // Tell our caller to go around again
}


////////////////////////////////////////////////////////////////////////
//
// SieveCandidates -- This function sieves a full block of candidates
//
////////////////////////////////////////////////////////////////////////

void SieveCandidates(int streamix)
{
  int threadsPerBlock = 256;
  int blocksPerGrid;

  // Initialize the sieve bit-map.  One thread per 32-bit word in the bitmap.
  blocksPerGrid = ((csieve_bits+31)/32 + threadsPerBlock - 1) / threadsPerBlock;

  // printf("Calling rcv_set_sieve_bits<<<%u,%u,%u,%u>>>(%u)\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream, csieve_bits);
  rcv_set_sieve_bits<<<blocksPerGrid, threadsPerBlock, 0, context[streamix].stream>>> (
        csieve_bits,          /* number of bits in bit-map */
        d_bitmapw[streamix]); /* bitmap for the sieve, 32-bit words */

  // tidoffseta and tidoffsetz range across the total represented in the root of the tree.
  unsigned int tidoffseta;    // offset (from threadId 0) to the first thread we plan to launch
  unsigned int tidoffsetz;    // offset (from threadId 0) just past the last thread we plan to launch
                              // (tidoffsetz-tidoffseta is the number of threads we plan to launch)
  unsigned int i;

  // Start sieving the primes...
  for(tidoffseta = 0, i=0; tidoffseta < h_ktree[1]; tidoffseta = tidoffsetz, i+=1)
  {
    tidoffsetz = tidoffseta + h_kncount[i];  // Assume we're going to sieve one prime.


    if (h_primep[i] == 61*1)  // Very special handling when we hit this prime
    {
      // One thread per 32-bit word in the bitmap.
      blocksPerGrid = ((csieve_bits+31)/32 + threadsPerBlock - 1) / threadsPerBlock;
      // printf("Calling rcv_sieve_small_13_61<<<%u,%u,%u,%u>>>\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream);
      rcv_sieve_small_13_61<<<blocksPerGrid, threadsPerBlock, 0, context[streamix].stream>>> (
            ckstart,           /* lowest k-value in current sieve */
            csieve_bits,       /* number of bits in current sieve */
            &d_bdelta[streamix][i-12],   /* pass pointer to 13 consecutive deltas from p=13 to p=61 */
            d_bitmapw[streamix]);        /* bitmap for the sieve, 32-bit words */
    }

    if (h_primep[i] == 127*1)  // Very special handling when we hit this prime
    {
      // One thread per 64-bit word in the bitmap.
      blocksPerGrid = ((csieve_bits+63)/64 + threadsPerBlock - 1) / threadsPerBlock;
      // printf("Calling rcv_sieve_small_67_127<<<%u,%u,%u,%u>>>\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream);
      rcv_sieve_small_67_127<<<blocksPerGrid, threadsPerBlock, 0, context[streamix].stream>>> (
            ckstart,           /* lowest k-value in current sieve */
            csieve_bits,       /* number of bits in current sieve */
            &d_bdelta[streamix][i-12],   /* pass pointer to 13 consecutive deltas from p=67 to p=127 */
            d_bitmapw[streamix]);        /* bitmap for the sieve, 32-bit words */
    }

    if (h_primep[i] == 251*1)  // Very special handling when we hit this prime
    {
      // One thread per 128-bit word in the bitmap.
      blocksPerGrid = ((csieve_bits+127)/128 + threadsPerBlock - 1) / threadsPerBlock;
      // printf("Calling rcv_sieve_small_131_251<<<%u,%u,%u,%u>>>\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream);
      rcv_sieve_small_131_251<<<blocksPerGrid, threadsPerBlock, 0, context[streamix].stream>>> (
            ckstart,           /* lowest k-value in current sieve */
            csieve_bits,       /* number of bits in current sieve */
            &d_bdelta[streamix][i-22],   /* pass pointer to 23 consecutive deltas from p=131 to p=251 */
            d_bitmapw[streamix]);        /* bitmap for the sieve, 32-bit words */
    }

    if (h_primep[i] == 509*1)  // Very special handling when we hit this prime
    {
      // One thread per 256-bit word in the bitmap.
      // Here, we use half the normal threadsPerBlock, due to large shared memory usage
      blocksPerGrid = ((csieve_bits+255)/256 + (threadsPerBlock/2) - 1) / (threadsPerBlock/2);
      // printf("Calling rcv_sieve_small_257_509<<<%u,%u,%u,%u>>>\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream);
      rcv_sieve_small_257_509<<<blocksPerGrid, threadsPerBlock/2, 0, context[streamix].stream>>> (
            ckstart,           /* lowest k-value in current sieve */
            csieve_bits,       /* number of bits in current sieve */
            &d_bdelta[streamix][i-42],   /* pass pointer to 43 consecutive deltas from p=257 to p=509 */
            d_bitmapw[streamix]);        /* bitmap for the sieve, 32-bit words */
    }

    if (h_primep[i] == 1021*1)  // Very special handling when we hit this prime
    {
      // One thread per 512-bit word in the bitmap.
      // Here, we use half the normal threadsPerBlock, due to large shared memory usage
      blocksPerGrid = ((csieve_bits+255)/256 + (threadsPerBlock/2) - 1) / (threadsPerBlock/2);
      // printf("Calling rcv_sieve_small_521_1021<<<%u,%u,%u,%u>>>\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream);
      rcv_sieve_small_521_1021<<<blocksPerGrid, threadsPerBlock/2, 0, context[streamix].stream>>> (
            ckstart,           /* lowest k-value in current sieve */
            csieve_bits,       /* number of bits in current sieve */
            &d_bdelta[streamix][i-74],   /* pass pointer to 75 consecutive deltas from p=521 to p=1021 */
            d_bitmapw[streamix]);        /* bitmap for the sieve, 32-bit words */
      }

    if (h_primep[i] == 2039*1)  // Very special handling when we hit this prime
    {
      // One thread per 1024-bit word in the bitmap.
      // Here, we use 1/8 the normal threadsPerBlock, due to large shared memory usage
      blocksPerGrid = ((csieve_bits+255)/256 + (threadsPerBlock/8) - 1) / (threadsPerBlock/8);
      // printf("Calling rcv_sieve_small_1031_2039<<<%u,%u,%u,%u>>>\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream);
      rcv_sieve_small_1031_2039<<<blocksPerGrid, threadsPerBlock/8, 0, context[streamix].stream>>> (
            ckstart,           /* lowest k-value in current sieve */
            csieve_bits,       /* number of bits in current sieve */
            &d_bdelta[streamix][i-136],   /* pass pointer to 137 consecutive deltas from p=1031 to p=2039 */
            d_bitmapw[streamix]);         /* bitmap for the sieve, 32-bit words */
      }

    if (h_primep[i] > 2048)                          // Use the general-purpose siever?
    {
      // Once we're to the general-purpose siever, we have a choice whether to sieve
      // one prime at a time, in which case atomic accesses to the sieve bits is not
      // required.  Or we can sieve everything that's left.  Since the above special-
      // purpose kernels handled all of the very small primes, our best strategy is
      // usually to just sieve all that remains via a single kernel.
     
      if ((tidoffsetz-tidoffseta) < threadsPerBlock*6)  // Small primes, with many threads, sieve prime-by-prime
        tidoffsetz = h_ktree[1];  // Large primes, with few threads, run helter-skelter.

      blocksPerGrid = (tidoffsetz-tidoffseta + threadsPerBlock - 1) / threadsPerBlock;
      // printf("Calling rcv_sieve_primes<<<%u,%u,%u,%u>>>\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream);
      rcv_sieve_primes<<<blocksPerGrid, threadsPerBlock, 0, context[streamix].stream>>> (
          tidoffseta,         /* Offset from tid to first tid's element in tree */
          tidoffsetz,         /* Offset from tid to just past last tid's element in tree */
          d_primep,           /* pointer to list of primes */
          MAX_SIEVE_PRIMES,   /* number of elements in list */
          ckstart,            /* lowest k-value in this sieve */
          csieve_bits,        /* number of bits in this sieve */
          d_bdelta[streamix], /* pointer to starting deltas per prime */
          d_kncount,          /* pointer to count of k-values per prime */
          d_ktree,            /* pointer to tree of count of k-values per prime */
          d_bitmapw[streamix]);/* bitmap for the sieve, 32-bit words */
    }

  }
}

////////////////////////////////////////////////////////////////////////
//
// LinearizeCandidates -- This function converts the sieved bits to a linear list
//
////////////////////////////////////////////////////////////////////////

void LinearizeCandidates(int streamix)
{
  int width;
  int threadsPerBlock;
  int blocksPerGrid;


  // Reset atomic index into linear array
  width = AX_COLUMNS;
  threadsPerBlock=(width+31) & ~31;  /* round up to warp size of 32 */
  blocksPerGrid = (width + threadsPerBlock - 1) / threadsPerBlock;

  // printf("Calling rcv_reset_atomic_indexes<<<%u,%u>>>\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream);
  rcv_reset_atomic_indexes<<<blocksPerGrid, threadsPerBlock, 0, context[streamix].stream>>> (
        width,                   /* width of atomic index array */
        d_xaindexes[streamix]);  /* pointer to our atomic index array */
  

  // Convert bitmap to a list of candidates.  One thread per 32-bit word in the bitmap.
  threadsPerBlock = 256;
  blocksPerGrid = ((csieve_bits+31)/32 + threadsPerBlock - 1) / threadsPerBlock;

  // One thread per 32-bit word in the bitmap.
  // printf("Calling rcv_linearize_sieve<<<%u,%u>>>\n", blocksPerGrid, threadsPerBlock, 0, context[streamix].stream);
  rcv_linearize_sieve<<<blocksPerGrid, threadsPerBlock, 0, context[streamix].stream>>> (
        ckstart,                /* lowest k-value in current sieve */
        csieve_bits,            /* number of bits in current sieve */
        d_bitmapw[streamix],    /* bitmap for the sieve, 32-bit words */
        d_karray[streamix],     /* linear array of k-values */
        MAX_TF_IN_SIEVE,        /* number of spots in output array */
        d_xaindexes[streamix]); /* atomic index into karray */

#if 1
  // Copy final count of candidates back to local storage
  checkCudaErrors(hipMemcpyAsync(
          h_xaindexes[streamix],
          d_xaindexes[streamix],
          sizeof(*h_xaindexes[streamix])*AX_COLUMNS,
          hipMemcpyDeviceToHost,
          context[streamix].stream));
#endif
}

////////////////////////////////////////////////////////////////////////
//
// TrialFactorCandidates -- This function trial factors a list of candidates
//
////////////////////////////////////////////////////////////////////////

void TrialFactorCandidates(unsigned int exp, unsigned int kclass, int96 ckstart, unsigned int ncand, int streamix)
{
  unsigned long cbstart;

  cbstart = ((0llu + context[streamix].cbstart.d1)<<32) + context[streamix].cbstart.d0;

  // This is just a stub.  Call the actual trial factoring kernel, here!

}

////////////////////////////////////////////////////////////////////////
//
// Fetch bdelta array for debugging
//
////////////////////////////////////////////////////////////////////////

void FetchBdelta(int streamix)
{
  checkCudaErrors(hipMemcpyAsync(
          h_bdelta[streamix],
          d_bdelta[streamix],
          sizeof(*h_bdelta[streamix])*MAX_SIEVE_PRIMES,  // Specify the number of bytes in the array
          hipMemcpyDeviceToHost,
          context[streamix].stream));    // Begin fetching delta array using specified stream
}


////////////////////////////////////////////////////////////////////////
//
// Debug the bdelta array
//
////////////////////////////////////////////////////////////////////////

void DebugBdelta(int streamix)
{
  bool bSanityCheck = true;    // When set, we sanity-check the array

  // Caller is responsible for synchronization to be certain our buffer is ready to go

  if (bSanityCheck)
  {
    unsigned long cbstart;     // Where the current window into the infinite bitmap begins
    cbstart = ((0llu + context[streamix].cbstart.d1)<<32) + context[streamix].cbstart.d0;

    // For each prime in our sieve, confirm that the bdelta value is correct
    for (unsigned int i=0; i<MAX_SIEVE_PRIMES; i+=1)
    {
      unsigned int f;
      unsigned int bdelta;

      f = h_primep[i];                 // Get the current prime
      bdelta = h_bdelta[streamix][i];  // Get delta (from the start of the bitmap) for current prime

      // Sieving is meaningless when the small prime is equal to the Mersenne exponent
      if (context[streamix].exp == f)  // Is the Mersenne exponent equal to this small prime?
        if (bdelta < SIEVE_BITS)       // bdelta must be large, so we don't do any sieving with this small prime
          printf("BDelta[%7u]:  prime=%7u == Mersenne exponent;  bdelta=%7u < MAX_SIEVE_PRIMES=%7u\n", i, f, bdelta, SIEVE_BITS);

      // For the other 99.99% of the cases, our sieve prime is relatively prime to the Mersenne exponent
      if (context[streamix].exp != f)  // Normally, the Mersenne exponent is not a small prime
      {
        if (bdelta >= f)               // delta to first sievable bit must be in the range [0,f)
          printf("BDelta[%7u]:  prime=%7u <= bdelta=%7u.  (Delta too large)\n", i, f, bdelta);

        unsigned long kmodf;
        unsigned long qmodf;

        kmodf  = 4620llu * ((cbstart + bdelta) % f) + context[streamix].kclass;
        kmodf %= f;
        qmodf  = 2llu * kmodf * (context[streamix].exp % f) + 1;
        qmodf %= f;
        if (qmodf != 0)
          printf("dbd:  prime=%7u; bdelta=%7u.  (At bdelta, q mod prime = %7u != 0.)\n", f, bdelta, qmodf);

      }
    }
  }
}


////////////////////////////////////////////////////////////////////////
//
// Fetch bitmap for debugging
//
////////////////////////////////////////////////////////////////////////

void FetchBitmap(int streamix)
{
  checkCudaErrors(hipMemcpyAsync(
          h_bitmapw[streamix],
          d_bitmapw[streamix],
          (context[streamix].csieve_bits+31)/32 * 4,  // Specify the number of sieve bytes used
          hipMemcpyDeviceToHost,
          context[streamix].stream));    // Begin fetching bitmap using specified stream
}


////////////////////////////////////////////////////////////////////////
//
// Debug the bitmap
//
////////////////////////////////////////////////////////////////////////

void DebugBitmap(int streamix)
{
  bool bDebug = false;             // When set, we show some of the bitmap
  bool bCountbits = true;          // When set, we count how many survivors are present

  // Caller is responsible for synchronization to be certain our buffer is ready to go
  if (bDebug)
  {
    // Display first 64 bytes of sieve bitmap
    for (int i=0; i<64; i+=1)
    {
      if (i%16 == 0)
        printf("+%6.6X  ", i);
      printf("%2.2X ", ((unsigned char *)h_bitmapw[streamix])[i] );
      if (i%16 == 15)
        printf("\n");
    }
    printf("...\n");

    // Display last 64 bytes of sieve bitmap
    for (int i=4*((context[streamix].csieve_bits+31)/32)-64; i< 4*((context[streamix].csieve_bits+31)/32); i+=1)
    {
      if (i%16 == 0)
        printf("+%6.6X  ", i);
      printf("%2.2X ", ((unsigned char *)h_bitmapw[streamix])[i] );
      if (i%16 == 15)
        printf("\n");
    }
  }

  if (bCountbits)
  {
    // Count the number of bits set in the bitmap
    int t = 0;
    for (unsigned int i=0; i<context[streamix].csieve_bits; i+=1)    // Count all bits still set
      if (h_bitmapw[streamix][i>>5]&(1<<(i&31)))
      {
        t +=1;
      }
    printf("%9u bits found\n", t);
  }

  //////////////////////////////////////////////////////////////////////////
  // Here, we can inspect the bitmap in h_bitmapw.  Perhaps for part of a
  // validation suite.  Perhaps for a specific debugging issue.  As long
  // as we use atomic updates in the rcv_sieve_bitmap kernel, the contents
  // of this bitmap should be repeatable.  If we don't use atomic updates
  // in the rcv_sieve_bitmap kernel, we may see some extraneous 1-bits.
  // We should *never* see any extraneous 0-bits!
  //////////////////////////////////////////////////////////////////////////

}

////////////////////////////////////////////////////////////////////////
//
// Fetch candidates for debugging
//
////////////////////////////////////////////////////////////////////////

void FetchCandidates(int streamix)
{
  // Get atomic index, which is essentially the number of candidates in the linearized list.
  // printf("h_xaindexes[i]=%16.16lX, d_xaindexes[i]=%16.16lX, size=%u, stream=%u\n",
  //         h_xaindexes[streamix], d_xaindexes[streamix], sizeof(*h_xaindexes[streamix])*AX_COLUMNS, context[streamix].stream);
  checkCudaErrors(hipMemcpyAsync(
          h_xaindexes[streamix],
          d_xaindexes[streamix],
          sizeof(*h_xaindexes[streamix])*AX_COLUMNS,
          hipMemcpyDeviceToHost,
          context[streamix].stream));  // Begin fetching the atomic index, which contains #candidates

  // Note.  If we wanted to wait for the above to complete, we could fetch
  // just the used portion of the list, below.  We don't want to wait, so we
  // fetch the entire list.

  // Get the full linear list.  [We even fetch beyond the limits given by h_xaindexes[0].]
  // printf("h_karray[i]=%16.16lX, d_karray[i]=%16.16lX, size=%u, stream=%u\n",
  //         h_karray[streamix], d_karray[streamix], sizeof(*h_karray[streamix])*MAX_TF_IN_SIEVE, context[streamix].stream);
  checkCudaErrors(hipMemcpyAsync(
          h_karray[streamix],
          d_karray[streamix],
          sizeof(*h_karray[streamix])*MAX_TF_IN_SIEVE,
          hipMemcpyDeviceToHost,
          context[streamix].stream));  // Begin fetching full candidate list
}


////////////////////////////////////////////////////////////////////////
//
// Debug candidates
//
////////////////////////////////////////////////////////////////////////

void DebugCandidates(int streamix)
{
  bool bShowstats = true;          // When set, we report how many candidates were allocated, and from what pool
  bool bShowsample = false;        // When set, we show one sample candidate
  bool bCompareBitmap = true;      // When set, each candidate will be checked against the bitmap and vice-versa
  bool bMissedSieve = false;       // When set, is trial factored with all of our factors.  [Very slow.]

#ifdef DEBUGBITMAP
#else
  bCompareBitmap = false;          // We *cannot* compare candidates to the bitmap if we don't have it!
#endif

  // Caller is responsible for synchronization to be certain our buffers are ready to go

  if (bShowstats)
  {
    printf("%9u candidates found, %9u candidates tested, %6.4f survived sieve of %u primes >= 13.\n",
            h_xaindexes[streamix][0], context[streamix].csieve_bits, (h_xaindexes[streamix][0]*1.0)/(context[streamix].csieve_bits*1.0), MAX_SIEVE_PRIMES);
  }

  ///////////////////////////////////////////////////////////////////////////////////
  // Here, we inspect the linearized list of candidates, in h_karray.  Perhaps
  // for part of a validation suite.  Perhaps for a specific debugging issue.
  // Remember, however, the list is not guaranteed to be in order, since kernel
  // blocks run independently of each other and CUDA does not guarantee their
  // execution order.
  ///////////////////////////////////////////////////////////////////////////////////

  if (bCompareBitmap)             // Extensive test that Candidate List is a perfect match to bitmap?
  {
    bool bError = false;          // Once an error occurs, the next phase is aborted.
    unsigned long cbstart;
    cbstart = ((0llu + context[streamix].cbstart.d1)<<32) + context[streamix].cbstart.d0;

    //
    // Phase I.  Check that every surviving Candidate is in the bitmap.
    //
    for (unsigned int i=0; i<h_xaindexes[streamix][0]; i+=1)
    {
      unsigned int b;

      b = h_karray[streamix][i];  // Get the next candidate

      if (b > context[streamix].csieve_bits)  // Is our b offset within bounds of bitmap?
      {
        printf("Candidate %u:  b=%u, B=%llu exceeds bounds of bitmap\n", i, b, cbstart+b);
        bError=true;
        continue;                            // Avoid blowing the array bounds
      }

      if (h_bitmapw[streamix][b>>5]&(1<<(b&31)))  // Is the corresponding bit set in the bitmap?
        ;
      else
      {
        printf("Candidate %u:  b=%u, B=%llu, k=4620*B+%u, q=2*k*%u+1 is in linear list, but not in bitmap\n",
                             i, b, cbstart+b, context[streamix].kclass, context[streamix].exp);
        bError=true;
      }
    }

    //
    // Phase II.  Check for duplicate candidates.
    //            *** WARNING *** THIS IS A DESTRUCTIVE TEST OF THE BITMAP
    //            If you need the bitmap after this, you better save a copy.
    //
    if (!bError)
        for (unsigned int i=0; i<h_xaindexes[streamix][0]; i+=1)
    {
      unsigned int b;

      b = h_karray[streamix][i];  // Get the next candidate

      if (h_bitmapw[streamix][b>>5]&(1<<(b&31)))  // Is the corresponding bit set in the bitmap?
        h_bitmapw[streamix][b>>5] &= ~(1<<(b&31));   // turn off one bit
      else
      {
        printf("Candidate %u:  b=%u, B=%llu, k=4620*B+%u, q=2*k*%u+1 is duplicated in linear list.\n",
                             i, b, cbstart+b, context[streamix].kclass, context[streamix].exp);
        bError=true;
      }
    }

    //
    // Phase III.  Check for candidates in the bitmap that weren't in the linear list
    //
    if (!bError)
    {
      // Count the number of bits remaining in the bitmap
      int t = 0;
      for (unsigned int b=0; b<context[streamix].csieve_bits; b+=1)    // Count all bits still set
        if (h_bitmapw[streamix][b>>5]&(1<<(b&31)))
        {
          t +=1;

          printf("Candidate:  b=%u, B=%llu, k=4620*B+%u, q=2*k*%u+1 is in bitmap, but not in linear list.\n",
                              b, cbstart+b, context[streamix].kclass, context[streamix].exp);
        }
      if (t != 0)
        printf("%u candidates are present in the bitmap, but not in linear list.\n", t);
    }
  }

  if (bShowsample)        // Display first candidate from this batch?
  {
    unsigned long cbstart;
    cbstart = ((0llu + context[streamix].cbstart.d1)<<32) + context[streamix].cbstart.d0;

    unsigned int i;
    unsigned int b;
    i = 0;                // First candidate is as good a sample as any.
    b = h_karray[streamix][i];  // Get the next candidate

    printf("Candidate %u:  b=%u, B=%llu, k=4620*B+%u, q=2*k*%u+1\n",
                       i, b, cbstart+b, context[streamix].kclass, context[streamix].exp);
  }

  // Following is very slow, as MAX_SIEVE_PRIMES increases
  if (bMissedSieve)       // Trial factor each candidate with all of our small primes?
  {
    unsigned long cbstart;
    cbstart = ((0llu + context[streamix].cbstart.d1)<<32) + context[streamix].cbstart.d0;

    for (unsigned int i=0; i<h_xaindexes[streamix][0]; i+=1)
    {
      unsigned int b;
      b = h_karray[streamix][i];  // Get the next candidate

      // Check for prime divisors the old-fashioned way
      for (int j=0; j<MAX_SIEVE_PRIMES; j+=1)
      {
        unsigned int f;
        unsigned long long k;
        unsigned long long q;
        f  = h_primep[j];
        k  = 4620llu * ((cbstart + b) % f) + context[streamix].kclass;
        k %= f;
        q  = 2llu * k * context[streamix].exp + 1;
        q %= f;
        if (q == 0llu)
        {
          printf("Candidate %u:  b=%u, B=%llu, k=4620*B+%u, q=2*k*%u+1, is divisible by %6u\n",
                           i, b, cbstart+b, context[streamix].kclass, context[streamix].exp, f);
          break;
        }
      }
    }
  }

}


int main(void)
{

  int streamix;                     // Current stream to schedule
  int laggingix;                    // Current stream to be debriefed
  int nLinearized;                  // Number of streams for which linearized event is scheduled

  bool bDebugBdelta;
  bool bDebugBitmap;
  bool bDebugCandidates;

#ifdef DEBUGBDELTA
  bDebugBdelta = true;    // When set, permits examination of the delta array
#else
  bDebugBdelta = false;   // When not set, saves copying large array from device
#endif

#ifdef DEBUGBITMAP
  bDebugBitmap = true;    // When set, permits examination of the sieve bitmap
#else
  bDebugBitmap = false;   // When not set, saves copying large bitmap from device
#endif

#ifdef DEBUGCANDIDATES
  bDebugCandidates = true;  // When set, permits examination of the candidate list
#else
  bDebugCandidates = false; // When not set, saves copying large candidate list from device
#endif

  InitApplication();      // Go perform application initialization

  // Initialize a set of stream-handles and events to keep things flowing
  for (int i=0; i<MAX_STREAMS; i+=1)
  {
    checkCudaErrors( hipStreamCreate(&context[i].stream) );
    checkCudaErrors( hipEventCreateWithFlags(&context[i].linearized_event,  hipEventBlockingSync) );
  }
  streamix = 0;                 // First stream to be utilized
  nLinearized = 0;              // No streams have reached LinearizedCandidates, yet


  ////////////////////////////////////////////////////////////////////
  // This data should come from a "worktodo" file.  Hard-coded now,
  // just for prototyping purposes.
  ////////////////////////////////////////////////////////////////////
  
  InitMersenne(MERSENNE_EXPONENT, QBITMIN, QBITMAX);  // Go perform initialization for a new Mersenne number

  for (int kclass=MINKCLASS; kclass<=MAXKCLASS; kclass+=1)
  {
    bool bGoodClass;

    bGoodClass = TestClass(MERSENNE_EXPONENT, kclass); // If class contains no primes, returns false
    if (bGoodClass)
    {
      InitClass(MERSENNE_EXPONENT, kclass);  // Go perform initialization for a new class
      bool bMoreToSieve;
      bMoreToSieve = true;

      // Loop as long as we have more work to enqueue or we have work left to dequeue
      while (bMoreToSieve || nLinearized != 0)
      {
        if (bMoreToSieve)
        {
          PrepClassBatch(MERSENNE_EXPONENT, kclass, streamix);  // Setup for the next sieve block
          if (bDebugBdelta)                  // Do we want to fetch and examine the delta array?
            FetchBdelta(streamix);             // For debugging, go initiate fetching of the array
          SieveCandidates(streamix);         // Sieve the candidates in current sieve block
          if (bDebugBitmap)                  // Do we want to fetch and examine the bitmap?
            FetchBitmap(streamix);             // For debugging, go initiate fetching of the bitmap
          LinearizeCandidates(streamix);     // Extract candidates from bitmap to a list
          if (bDebugCandidates)              // Are we configured to fetch and examine the candidate list?
            FetchCandidates(streamix);         // For debugging, go initiate fetching of the candidates
          checkCudaErrors( hipEventRecord(context[streamix].linearized_event, context[streamix].stream) );
                                             // record event in this stream
          if (nLinearized == 0)              // Have we scheduled our first EventRecord?
            laggingix = streamix;            // Yes.  Remember oldest stream scheduled
          nLinearized += 1;                  // Bump the number of linearized_event kernels running
        }

        // Wait 'til all streams are performing sieving to begin factoring.
        // But if this class is out of sieving, empty the pipeline.
        if (nLinearized == MAX_STREAMS ||
            (!bMoreToSieve && nLinearized != 0))    // Wait 'til all streams are performing sieving to begin factoring
        {
          checkCudaErrors( hipEventSynchronize(context[laggingix].linearized_event) );
                                           // Wait for oldest kernel's stream to finish linearizing a block
          nLinearized -= 1;                // One stream has caught up

          if (bDebugBdelta)                // Are we configured to fetch and examine the delta array?
            DebugBdelta(laggingix);          // For debugging, go analyze the delta array
          if (bDebugBitmap)                // Are we configured to fetch and examine the bitmap?
            DebugBitmap(laggingix);          // For debugging, go analyze the bitmap
          if (bDebugCandidates)            // Are we configured to fetch and examine the candidate list?
            DebugCandidates(laggingix);      // For debugging, go analyze the candidate list

          TrialFactorCandidates(
                      context[laggingix].exp,
                      context[laggingix].kclass,
                      context[laggingix].ckstart,
                      h_xaindexes[laggingix][0],
                      laggingix);   // Go trial factor this batch of candidates

          laggingix = (laggingix+1)%MAX_STREAMS; // Next stream to be dequeued
        }

        streamix = (streamix+1)%MAX_STREAMS;     // Next batch of sieves switches to the next buffer
        if (bMoreToSieve)
          bMoreToSieve = NextClassBatch(MERSENNE_EXPONENT, kclass);
                                   // If more work at this class, get it setup
      }
    }
  }

  ////////////////////////////////////////////////////////////////////
  // Here, we should loop back for more work from a "worktodo"
  // file.  For prototyping, we just run a single Factor=
  ////////////////////////////////////////////////////////////////////
  
  // Clean up our stream-handles and events
  for (int i=0; i<MAX_STREAMS; i+=1)
  {
    checkCudaErrors( hipStreamSynchronize(context[i].stream) );
    checkCudaErrors( hipEventDestroy(context[i].linearized_event) );
    checkCudaErrors( hipStreamDestroy(context[i].stream) );
  }

  hipDeviceSynchronize();


  FreeHostMemory(bPinGenericMemory, &h_m1, &h_primep,  sizeof(*h_primep)*MAX_SIEVE_PRIMES);
  h_primep = NULL;

  FreeHostMemory(bPinGenericMemory, &h_m3, &h_kncount, sizeof(*h_kncount)*MAX_SIEVE_PRIMES);
  h_kncount = NULL;

  FreeHostMemory(bPinGenericMemory, &h_m4, &h_ktree,   sizeof(*h_ktree)*(2*MAX_SIEVE_PRIMES));
  h_ktree = NULL;

  for (int i=0; i<MAX_STREAMS; i+=1)
  {
    FreeHostMemory(bPinGenericMemory, &h_m5, &h_bitmapw[i], sizeof(*h_bitmapw[i])*((SIEVE_BITS+31)/32));
    h_bitmapw[i] = NULL;

    FreeHostMemory(bPinGenericMemory, &h_m7, &h_karray[i],  sizeof(*h_karray[i] )*MAX_TF_IN_SIEVE);
    h_karray[i]  = NULL;

    FreeHostMemory(bPinGenericMemory, &h_m8, &h_xaindexes[i], sizeof(*h_xaindexes[i])*AX_COLUMNS);
    h_xaindexes[i]  = NULL;
  }

  hipFree(d_primep);
  d_primep = NULL;

  hipFree(d_kncount);
  d_kncount = NULL;

  hipFree(d_ktree);
  d_ktree = NULL;

  for (int i=0; i<MAX_STREAMS; i+=1)
  {
    hipFree(d_bdelta[i]);
    d_bdelta[i] = NULL;

    hipFree(d_bitmapw[i]);
    d_bitmapw[i] = NULL;

    hipFree(d_karray[i] );
    d_karray[i]  = NULL;

    hipFree(d_xaindexes[i] );
    d_xaindexes[i]  = NULL;
  }

  exit (0);
}



///////////////////////////////////////////////////////////////////
//
// rcv_init_class
//
//   Background:
//
//     Let  M(p) = 2^p-1  be a Mersenne number.
//     By theorem, all prime factors of M(P) are of the form
//          q = 2*k*p+1,
//     where k is an integer.
//
//     Since we are interested in finding the smallest
//     factors of Mersenne numbers, we are interested
//     in finding prime factors of Mersenne numbers.  No
//     such prime factor, q, will contain any small prime
//     factors.  Furthermore, by theorem, any prime
//     factor of a Mersenne number will be congruent to
//     +/-1 (mod 8).
//
//     What is a "class"?  A class is a set of numbers that have a
//     common remainder, modulo the product of a set of small primes.
//     In our implementation, there are 4620 = 2^2 * 3 * 5 * 7 * 11
//     possible classes.
//
//     All of the candidates we generate in a batch are of the same
//     class, so all those candidates we generate will have the same
//     remainder, mod 4620.
//
//     The reason is simply for performance.  If we never even
//     attempt to generate candidates that are a multiple of 3,
//     then we save 1/3 of the work we might otherwise perform.
//     If we never generate candidates that are a multiple of 5,
//     then we save another 1/5 of the work we might otherwise
//     have performed.  Ditto for multiples of 7 and 11.
//
//     Furthermore, all candidates must be equal to 1 or 7,
//     modulo 8 to be a factor of a Mersenne number.  This lets
//     us reduce by another factor of 2 the work we might
//     otherwise have performed.  [It might appear we should be
//     reducing the work by a factor of 4, but we are inherently
//     only considering odd candidates, so we can only
//     eliminate candidates that are equal to 3 or 5, modulo 8.]
//
//     By theorem, all factors of a mersenne number are of the
//     form, 2*k*p+1, where k is a positive integer and p is the
//     exponent of the Mersenne number.
//
//     The following table contains (2*k*p) mod 8 for each
//     possible combination of a k-value with an odd Mersenne exponent.
//
//                   (2*k) mod 8
//       p mod 8   0   -   2   -   4   -   6   -
//                -------------------------------
//          1   |  0   -   2   -   4   -   6   -
//          3   |  0   -   6   -   4   -   2   -
//          5   |  0   -   2   -   4   -   6   -
//          7   |  0   -   6   -   4   -   2   -
//
//     Next, we look at the candidates (2*k*p+1) mod 8.  The valid
//     candidates are equal to 1 or 7.  Others need not be generated.
//
//                   (2*k) mod 8
//       p mod 8   0   1   2   3   4   5   6   7
//                -------------------------------
//          1   |  1   -   -   -   -   -   7   -
//          3   |  1   -   7   -   -   -   -   -
//          5   |  1   -   -   -   -   -   7   -
//          7   |  1   -   7   -   -   -   -   -
//
//     This shows us that if (p mod 4) == 1, then k must equal 0 or 3 (modulo 4).
//                    and if (p mod 4) == 3, then k must equal 0 or 1 (modulo 4).
//
//     For a given Mersenne exponent, we only generate Candidates from among
//     960 of the possible 4620 classes:
//
//          2 * 4 * 6 * 10 * 2    960
//          ------------------ = ----
//          3 * 5 * 7 * 11 * 4   4620
//
//     Within each class, we sieve out Candidates that are multiples of higher
//     primes, 13, 17, 19, 23, etc.
//
//
//     By our own internal convention, we number the classes based on k (mod 4620),
//     as used in the formula q = 2*k*p+1.  And all candidates, q, generated with a batch
//     of a given class will be congruent to each other modulo 9240.  Note that
//     we will consider only 960 of the 9240 possible candidate residues.  Also
//     note that the candidate residues will normally not be our class number.
//
//
//
//
//   Input parameters:
//
//     exp:  The exponent of the Mersenne Number being tested.
//           E.g., to trial-factor 2^100099-1, exp=100099
//
//     class:  The "class" of factors about to be tested.  There
//             are 4620 = 2^2 * 3 * 5 * 7 * 11 possible classes.
//             All factors tested within a given class will have
//             the same remainder, mod 4620.  Among the possible
//             classes, 960 classes may contain prime factors of
//             the Mersenne number under test.
//
//             Among the 4620 classes, only 960 classes are not
//             divisible by one of the 4 small odd primes and
//             congruent to +/- 1 (modulo 8).
//
//              960 = 2 * 2 * 4 * 6 * 10
//             4620 = 8 * 3 * 5 * 7 * 11 / 2.
//             (Even numbers are inherently excluded by q=2kp+1,
//             and are not a member of any class.)
//
//     kdiv4620:  This is the starting value of k, divided by
//             4620, and truncated to an integer.  The starting
//             value for k is therefore equal to 4620*k + kdelta,
//             where 0 <= kdelta < 4620, such that k mod 4620 = class.
//
//       kdelta:  The offset (bits) from the first bit of the sieve
//                to the first bit to be sieved out.  [Thereafter,
//                every prime-th bit is sieved out.]  (For each prime,
//                0 <= kdelta_prime < prime.)  (For very large primes,
//                in which no sieving will occur in this set of bits,
//                this may be beyond the bits available for sieving.)
//
// q  = 2*k*p + 1
// q0 = 2*k0*p + 1
// k0 = class + 4620*c0       k0 = kstart.  Corresponds to first bit of bitmap.
// kn = class + 4620*(c0+i)   0 <= i < SIEVE_BITS.  Corresponds to successive bits of bitmap.
// qn = 2*kn*p + 1
// qn = 2*(class + 4620*(c0+i))*p + 1
// qn = 2*class*p + 9240*c0*p + 9240*i*p + 1
// qn = q0 + 9240*i*p
//                            delta is chosen as the smallest i such that qn is divisible by the
//                              corresponding sieve-prime and qn >= q0.
//                            ndelta is chosen as the smallest i such that qn is divisible by the
//
///////////////////////////////////////////////////////////////////

__device__ unsigned int mod32bit(unsigned int n, unsigned int d)
{
  return n%d;
}

__device__ unsigned int modularinverse(unsigned int n, unsigned int d)
{
  int x, y, lastx, lasty, q, t;
  x=0; y=1; lastx=1; lasty=0;
  while (d != 0)
  {
    q=n/d;                      // Floor(n/d)
    t=d; d=n%d; n=t;            // d=n mod d; n=lastd;
    t=x; x=lastx-q*x; lastx=t;
    t=y; y=lasty-q*y; lasty=t;
  }
  return(lastx);
}

__global__ void __launch_bounds__(256, 4) rcv_init_class(
        unsigned int exp,       // Mersenne exponent, p, of M(p) = 2^p-1 */
        unsigned int nclass,    // Number of classes  (must = 4620) */
        unsigned int kclass,    // Number of this class */
        int96        kstart,    // Starting k-value.  Must=class (mod nclass) */
        unsigned int *d_plist,  // In: Pointer to list of primes for sieving */
        unsigned int pcount,    // In: Number of primes in sieve list */
        unsigned int *d_bdelta  // Out: First bit, s.t. q is a multiple of the prime */
        )
{

  unsigned int i;
  i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < pcount)               // Excess threads do not participate
  {
    // Compute lowest possible k_d, s.t. q=2(k+4620*k_d)p+1 is divisible by our prime
    unsigned int q0mp;
    if (d_plist[i] > 46341)  // Do we have to use long arithmetic?
    {
      unsigned long long ksmp;  // kstart mod (current small prime)
      unsigned long long qsmp;  // qstart mod (current small prime)
      ksmp = (             kstart.d2) % d_plist[i];
      ksmp = ((ksmp<<32) + kstart.d1) % d_plist[i];
      ksmp = ((ksmp<<32) + kstart.d0) % d_plist[i];

      qsmp = (2*ksmp*(exp % d_plist[i]) + 1) % d_plist[i];
      q0mp = qsmp;
    }
    else
    {
      unsigned int ksmp;  // kstart mod (current small prime)
      ksmp = (              kstart.d2)             % d_plist[i];
      ksmp = ((ksmp<<16) + (kstart.d1>>16       )) % d_plist[i];
      ksmp = ((ksmp<<16) + (kstart.d1&0x0000ffff)) % d_plist[i];
      ksmp = ((ksmp<<16) + (kstart.d0>>16       )) % d_plist[i];
      ksmp = ((ksmp<<16) + (kstart.d0&0x0000ffff)) % d_plist[i];

      q0mp = (2*ksmp*(exp % d_plist[i]) + 1) % d_plist[i];  // 2*46340*46340+1 < 2^32
    }

    unsigned int p9240mp;    // (2*4620*exponent) mod (current small prime)
    p9240mp = (9240llu * (exp % d_plist[i])) % d_plist[i];

    if (p9240mp == 0)        // Did our thread's prime equal the Mersenne exponent?
      d_bdelta[i] = SIEVE_BITS;    // Yes, don't attempt to sieve this prime
    else
    {
      unsigned int j;
               int p9240mpinv;
      p9240mpinv = modularinverse(p9240mp, d_plist[i]);
      if (p9240mpinv < 0)
        p9240mpinv += d_plist[i];

      // primes can exceed 65535, so use 64-bit multiply
      j = (0llu+d_plist[i]-q0mp)*(0llu+p9240mpinv) % d_plist[i];
      d_bdelta[i] = j;
    }
  }
}


///////////////////////////////////////////////////////////////////
//
// rcv_build_prime_tree
//
//   This function receives the set of primes for which sieving
//   will be performed.
//
//   Input parameters:
//
//       kcount:  Number of bits in sieve table.
//
//       plist:   Pointer to a list of primes to sieve the k-values.
//
//       pcount:  Number of primes in the sieve-list.
//
//       kncount: The maximum number of sievable bits, per prime.
//
//                A---V  Above and below are the same.  One is a list.  One is a tree.
//
//       ktree:   ktree is an array of 2*pcount+1 elements.
//                Element [0] is unused.
//                Element [n] is the sum of [2n] and [2n+1].
//                Elements [pcount+1 to 2*pcount] are the
//                same as the kncount list.
//
//   Example:
//   kcount = 1048576
//   kstart = First k-value in upcoming sieve
//   plist->     13,   17,   19,   23,   29,   31,   ...,   3497743
//   kdelta->     7,   15,   16,    1,    0,   15,   ...,   1589450
//   knn->    80660,61681,55189,45591,36158,33826,   ...,         1
//
///////////////////////////////////////////////////////////////////

__global__ void rcv_build_prime_tree(
        unsigned int *d_plist,  /* pointer to list of primes */
        unsigned int pcount,    /* number of elements in list */
        unsigned int kcount,    /* maximum number of bits in upcoming sieves */
        unsigned int *d_kncount,/* pointer to list of counts of k-values per prime */
        unsigned int *d_ktree   /* pointer to tree of counts of k-values per prime */
        )
{
  unsigned int i;

  for (i=threadIdx.x; i<2*pcount; i+=blockDim.x)
    d_ktree[i] = 0;
  __syncthreads();      // Does this speed things?

  unsigned int pcountpow2;      /* next power of 2 >= pcount */

  for (pcountpow2=1; pcountpow2 < pcount; pcountpow2 = pcountpow2+pcountpow2)
    ;

  unsigned int ndeeper;         /* number of elements 1 level deeper in tree */
  ndeeper = pcount+pcount - pcountpow2;

#ifdef DEBUGTREE
#if (__CUDA_ARCH__ >= 200) && (CUDART_VERSION >= 4010)
  if (threadIdx.x == 0)
    printf("pcount = %u; pcountpow2 = %u; ndeeper = %u\n", pcount, pcountpow2, ndeeper);
#endif
#endif

  // When building the tree, we must work within a single block,
  // because blocks cannot synchronize with each other.  Since
  // this data isn't massively parallel, that's OK.  But we do
  // let multiple threads compute the differences.

  for (i=threadIdx.x; i<ndeeper; i+=blockDim.x)
  {
    d_kncount[i] = (kcount+d_plist[i]-1)/d_plist[i];  // Maximum bits to clear for this prime
    d_ktree[pcountpow2+i] = d_kncount[i];  // Copy from list to tree
  }

  __syncthreads();      // Does this speed things?

  for ( ; i<pcount; i+=blockDim.x)
  {
    d_kncount[i] = (kcount+d_plist[i]-1)/d_plist[i];  // Maximum bits to clear for this prime
    d_ktree[pcount+i-ndeeper] = d_kncount[i];  // Copy from list to tree
  }

  __syncthreads();      // Ensure all differences are tallied.

  // Let a single block build the tree
  if (blockIdx.x == 0)
  {
    i = pcount-1;
    while (blockDim.x < i/2)         // Can we put all threads to work?
    {
      unsigned int myi;
      myi = i - (blockDim.x-1) + threadIdx.x;
      d_ktree[myi] = d_ktree[2*myi] + d_ktree[2*myi+1];
      i = i - blockDim.x;
      __syncthreads();	// Warps may not be in lock-step, so sync up
    }

    // Let a single thread finish the tree
    if (threadIdx.x == 0)
      for ( ; i>0; i-=1)
        d_ktree[i] = d_ktree[2*i] + d_ktree[2*i+1];
  }
}


__global__ void rcv_set_sieve_bits(
        unsigned int kcount,    /* number of bits in upcoming sieve */
        unsigned int *d_bitmapw /* pointer to bitmap of 32-bit words */
        )
{
  /* One thread, per 32-bit word of bitmap, please */
  unsigned int i;
  i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i < (kcount>>5))	// Excess threads don't participate
    d_bitmapw[i] = 0xffffffff;
}


// For your visualization pleasure, These tables show how patterns of consecutive
// factors of "13" and "23" appear through consecutive 32-bit words.

// Patterns of locations of factors of "13" in a 32-bit word
// .....1............1............1  [0] = 0, 13, 26
// ...........1............1.......  39, 52
// ....1............1............1.  65, 78, 91
// ..........1............1........  104, 117
// ...1............1............1..  130, 143, 156
// .........1............1.........  169, 182
// ..1............1............1...  195, 208, 221
// ........1............1..........  234, 247
// .1............1............1....  260, 273, 286
// .......1............1...........  299, 312
// 1............1............1.....  325, 338, 351
// ......1............1............  364, 377
// ............1............1......  390, 403,
// .....1............1............1  [13] = 416, 429, 442     32*13 = 416

// Patterns of locations of factors of "23" in a 32-bit word
// ........1......................1  [0] = 0, 23
// .................1..............  46
// ...1......................1.....  69, 92
// ............1...................  115
// .....................1..........  138
// .......1......................1.  161, 184
// ................1...............  207
// ..1......................1......  230, 253
// ...........1....................  276
// ....................1...........  299
// ......1......................1..  322, 345
// ...............1................  368
// .1......................1.......  391, 414
// ..........1.....................  437
// ...................1............  460
// .....1......................1...  483, 506
// ..............1.................  529
// 1......................1........  552, 575
// .........1......................  598
// ..................1.............  621
// ....1......................1....  644, 667
// .............1..................  690
// ......................1.........  713
// ........1......................1  [23] = 736, 759   32*23 = 736


__global__ void __launch_bounds__(256, 4) rcv_sieve_small_13_61(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta13,  /* pointer to list of 13 deltas for primes 13 through 61 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        )
{
  unsigned int i;
  __shared__ unsigned int s_bdelta[13]; // Space for 13 k-values in shared (fast) memory
#define kdelta13  s_bdelta[0]
#define kdelta17  s_bdelta[1]
#define kdelta19  s_bdelta[2]
#define kdelta23  s_bdelta[3]
#define kdelta29  s_bdelta[4]
#define kdelta31  s_bdelta[5]
#define kdelta37  s_bdelta[6]
#define kdelta41  s_bdelta[7]
#define kdelta43  s_bdelta[8]
#define kdelta47  s_bdelta[9]
#define kdelta53  s_bdelta[10]
#define kdelta59  s_bdelta[11]
#define kdelta61  s_bdelta[12]

  // We let the first 13 threads of each thread block simultaneously transfer
  // kdelta values from global memory to shared memory
  if (threadIdx.x < 13)
    s_bdelta[threadIdx.x] = d_bdelta13[threadIdx.x];

  __syncthreads();

// BITSLL:  13: 04002001  17: 00020001
// BITSRR:  13: 00080040  17: 00008000

#define BITSLL13 (1 | 1<<(   13) | 1<<(   13+13))
#define BITSRR13 (    1<<(32-13) | 1<<(32-13-13))

#define BITSLL17 (1 | 1<<(   17))
#define BITSRR17 (    1<<(32-17))

#define BITSLL19 (1 | 1<<(   19))
#define BITSRR19 (    1<<(32-19))

#define BITSLL23 (1 | 1<<(   23))
#define BITSRR23 (    1<<(32-23))

#define BITSLL29 (1 | 1<<(   29))
#define BITSRR29 (    1<<(32-29))

#define BITSLL31 (1 | 1<<(   31))
#define BITSRR31 (    1<<(32-31))

  // One thread, per 32-bit word of bitmap should be launched for this kernel, please.)
  i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < ((kcount+31)>>5))  // Excess threads don't participate.
  {
    unsigned int j;
    unsigned int k;
    unsigned int mask;

    // The following handles primes 13 < p < 32.  These are the primes
    // that fit within one 32-bit word, that haven't been inherently
    // sieved out via the "classes" mechanism.
    // Since we are executing with one thread per 32-bit word, we are
    // guaranteed to find sieve bits in our word for each and every prime.
    // With p = 13, we will always find 2 or 3 bits to sieve.
    // With 16 < p < 32, we will always find 1 or 2 bits to sieve.

    j = (i * 32 + 13-1 - kdelta13) / 13;        // Lowest bit this thread can reach with 32-bit accesses
    k = kdelta13 + j*13;
    mask  = (BITSLL13<<(k&31));

    j = (i * 32 + 17-1 - kdelta17) / 17;
    k = kdelta17 + j*17;
    mask |= (BITSLL17<<(k&31));

    j = (i * 32 + 19-1 - kdelta19) / 19;
    k = kdelta19 + j*19;
    mask |= (BITSLL19<<(k&31));

    j = (i * 32 + 23-1 - kdelta23) / 23;
    k = kdelta23+ j*23;
    mask |= (BITSLL23<<(k&31));

    j = (i * 32 + 29-1 - kdelta29) / 29;
    k = kdelta29 + j*29;
    mask |= (BITSLL29<<(k&31));

    j = (i * 32 + 31-1 - kdelta31) / 31;
    k = kdelta31 + j*31;
    mask |= (BITSLL31<<(k&31));


    // The following handles primes, 32 < p < 64.
    // Since we are executing with one thread per 32-bit word, we will either
    // find one or zero bits to sieve.  There is some inherent inefficiency,
    // since we may go through the motions, without sieving anything.
    // Theoretically, this technique could be used for even larger primes,
    // but the inefficiencies rapidly grow.

    j = (i * 32 + 37-1 - kdelta37) / 37;
    k = kdelta37 + j*37;
    mask |= (k>>5)==i ? (1<<(k&31)) : 0;

    j = (i * 32 + 41-1 - kdelta41) / 41;
    k = kdelta41 + j*41;
    mask |= (k>>5)==i ? (1<<(k&31)) : 0;

    j = (i * 32 + 43-1 - kdelta43) / 43;
    k = kdelta43 + j*43;
    mask |= (k>>5)==i ? (1<<(k&31)) : 0;

    j = (i * 32 + 47-1 - kdelta47) / 47;
    k = kdelta47 + j*47;
    mask |= (k>>5)==i ? (1<<(k&31)) : 0;

    j = (i * 32 + 53-1 - kdelta53) / 53;
    k = kdelta53 + j*53;
    mask |= (k>>5)==i ? (1<<(k&31)) : 0;

    j = (i * 32 + 59-1 - kdelta59) / 59;
    k = kdelta59 + j*59;
    mask |= (k>>5)==i ? (1<<(k&31)) : 0;

    j = (i * 32 + 61-1 - kdelta61) / 61;
    k = kdelta61 + j*61;
    mask |= (k>>5)==i ? (1<<(k&31)) : 0;

    if (d_bitmapw[i] & mask)    // Are any of our bits still on?
    {
      d_bitmapw[i] &= ~mask;    // If yes, turn them off.  If no, save a memory write.
    }
  }
}


__global__ void __launch_bounds__(256, 4) rcv_sieve_small_67_127(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta67,  /* pointer to list of 13 deltas for primes 67 through 127 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        )
{
  unsigned int i;
  __shared__ unsigned int smap[512];  // Two 32-bit words per thread per block
  __shared__ unsigned int s_bdelta[13]; // Space for 13 k-values in shared (fast) memory
#define kdelta67  s_bdelta[0]
#define kdelta71  s_bdelta[1]
#define kdelta73  s_bdelta[2]
#define kdelta79  s_bdelta[3]
#define kdelta83  s_bdelta[4]
#define kdelta89  s_bdelta[5]
#define kdelta97  s_bdelta[6]
#define kdelta101 s_bdelta[7]
#define kdelta103 s_bdelta[8]
#define kdelta107 s_bdelta[9]
#define kdelta109 s_bdelta[10]
#define kdelta113 s_bdelta[11]
#define kdelta127 s_bdelta[12]

  // We let the first 13 threads of each thread block simultaneously transfer
  // kdelta values from global memory to shared memory
  if (threadIdx.x < 13)
    s_bdelta[threadIdx.x] = d_bdelta67[threadIdx.x];

  __syncthreads();

  // One thread, per 64-bit word of bitmap should be launched for this kernel, please.)
  i = blockDim.x * blockIdx.x + threadIdx.x;

  // All threads *must* participate, since they write each other's results to global memory
  {
    unsigned int j;
    unsigned int k;

    // The following handles primes, 64 < p < 128.
    // Since we are executing with one thread per 64-bit word, we will either
    // find one or zero bits to sieve.

    // The bits we sieve will be ORed into one of these two 32-bit words.
    smap[2*threadIdx.x  ] = 0;
    smap[2*threadIdx.x+1] = 0;

#define SIEVE_64_BIT(p, kdeltap) { \
    j = (i * 64 + p-1 - kdeltap) / p; \
    k = kdeltap + j*p; \
    if ((k>>6) == i) \
      smap[2*threadIdx.x+((k>>5)&1)] |= 1<<(k&31); \
    }

    SIEVE_64_BIT( 67, kdelta67);
    SIEVE_64_BIT( 71, kdelta71);
    SIEVE_64_BIT( 73, kdelta73);
    SIEVE_64_BIT( 79, kdelta79);
    SIEVE_64_BIT( 83, kdelta83);
    SIEVE_64_BIT( 89, kdelta89);
    SIEVE_64_BIT( 97, kdelta97);
    SIEVE_64_BIT(101, kdelta101);
    SIEVE_64_BIT(103, kdelta103);
    SIEVE_64_BIT(107, kdelta107);
    SIEVE_64_BIT(109, kdelta109);
    SIEVE_64_BIT(113, kdelta113);
    SIEVE_64_BIT(127, kdelta127);

    __syncthreads();                    // Make sure everybody has stored their results
    d_bitmapw[2*i-2*threadIdx.x+           threadIdx.x] &= ~smap[           threadIdx.x];
    d_bitmapw[2*i-2*threadIdx.x+blockDim.x+threadIdx.x] &= ~smap[blockDim.x+threadIdx.x];
  }
}


__global__ void __launch_bounds__(256, 4) rcv_sieve_small_131_251(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta131,  /* pointer to list of 23 deltas for primes 131 through 251 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        )
{
  unsigned int i;
  __shared__ unsigned int smap[4*256];  // Four 32-bit words per thread per block
  __shared__ unsigned int s_bdelta[23]; // Space for 23 k-values in shared (fast) memory
#define kdelta131 s_bdelta[0]
#define kdelta137 s_bdelta[1]
#define kdelta139 s_bdelta[2]
#define kdelta149 s_bdelta[3]
#define kdelta151 s_bdelta[4]
#define kdelta157 s_bdelta[5]
#define kdelta163 s_bdelta[6]
#define kdelta167 s_bdelta[7]
#define kdelta173 s_bdelta[8]
#define kdelta179 s_bdelta[9]
#define kdelta181 s_bdelta[10]
#define kdelta191 s_bdelta[11]
#define kdelta193 s_bdelta[12]
#define kdelta197 s_bdelta[13]
#define kdelta199 s_bdelta[14]
#define kdelta211 s_bdelta[15]
#define kdelta223 s_bdelta[16]
#define kdelta227 s_bdelta[17]
#define kdelta229 s_bdelta[18]
#define kdelta233 s_bdelta[19]
#define kdelta239 s_bdelta[20]
#define kdelta241 s_bdelta[21]
#define kdelta251 s_bdelta[22]

  // We let the first 23 threads of each thread block simultaneously transfer
  // kdelta values from global memory to shared memory
  if (threadIdx.x < 23)
    s_bdelta[threadIdx.x] = d_bdelta131[threadIdx.x];

  __syncthreads();

  // One thread, per 128-bit word of bitmap should be launched for this kernel, please.)
  i = blockDim.x * blockIdx.x + threadIdx.x;

  // All threads *must* participate, since they write each other's results to global memory
  {
    unsigned int j;
    unsigned int k;

    // The following handles primes, 128 < p < 256.
    // Since we are executing with one thread per 128-bit word, we will either
    // find one or zero bits to sieve per thread.

    // The bits we sieve will be ORed into one of these four 32-bit words.
    smap[4*threadIdx.x  ] = 0;
    smap[4*threadIdx.x+1] = 0;
    smap[4*threadIdx.x+2] = 0;
    smap[4*threadIdx.x+3] = 0;

#define SIEVE_128_BIT(p, kdeltap) { \
    j = (i * 128 + p-1 - kdeltap) / p; \
    k = kdeltap + j*p; \
    if ((k>>7) == i) \
      smap[4*threadIdx.x+((k>>5)&3)] |= 1<<(k&31); \
    }

    SIEVE_128_BIT(131, kdelta131);
    SIEVE_128_BIT(137, kdelta137);
    SIEVE_128_BIT(139, kdelta139);
    SIEVE_128_BIT(149, kdelta149);
    SIEVE_128_BIT(151, kdelta151);
    SIEVE_128_BIT(157, kdelta157);
    SIEVE_128_BIT(163, kdelta163);
    SIEVE_128_BIT(167, kdelta167);
    SIEVE_128_BIT(173, kdelta173);
    SIEVE_128_BIT(179, kdelta179);
    SIEVE_128_BIT(181, kdelta181);
    SIEVE_128_BIT(191, kdelta191);
    SIEVE_128_BIT(193, kdelta193);
    SIEVE_128_BIT(197, kdelta197);
    SIEVE_128_BIT(199, kdelta199);
    SIEVE_128_BIT(211, kdelta211);
    SIEVE_128_BIT(223, kdelta223);
    SIEVE_128_BIT(227, kdelta227);
    SIEVE_128_BIT(229, kdelta229);
    SIEVE_128_BIT(233, kdelta233);
    SIEVE_128_BIT(239, kdelta239);
    SIEVE_128_BIT(241, kdelta241);
    SIEVE_128_BIT(251, kdelta251);

// Which strategy to write results to global memory?  atomicAnd isn't really necessary, as long
// as all kernel's prior to this one have finished before our kernel runs.
// Scrambling the copies from shared memory to global memory yields ~100% global load/store efficiency,
// but doesn't actually have much performance gain.
// Straightforward copy reports 25% global load/store efficiency, but isn't much slower.

    __syncthreads();    // Make sure everybody has stored their results
    d_bitmapw[4*i-4*threadIdx.x+             threadIdx.x] &= ~smap[             threadIdx.x];
    d_bitmapw[4*i-4*threadIdx.x+  blockDim.x+threadIdx.x] &= ~smap[  blockDim.x+threadIdx.x];
    d_bitmapw[4*i-4*threadIdx.x+2*blockDim.x+threadIdx.x] &= ~smap[2*blockDim.x+threadIdx.x];
    d_bitmapw[4*i-4*threadIdx.x+3*blockDim.x+threadIdx.x] &= ~smap[3*blockDim.x+threadIdx.x];
  }
}

__global__ void __launch_bounds__(128, 4) rcv_sieve_small_257_509(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta257,  /* pointer to list of 43 deltas for primes 257 through 509 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        )
{
  unsigned int i;
  __shared__ unsigned int smap[8*128];  // Eight 32-bit words per thread per block
  __shared__ unsigned int s_bdelta[43]; // Space for 43 k-values in shared (fast) memory
#define kdelta257 s_bdelta[0]
#define kdelta263 s_bdelta[1]
#define kdelta269 s_bdelta[2]
#define kdelta271 s_bdelta[3]
#define kdelta277 s_bdelta[4]
#define kdelta281 s_bdelta[5]
#define kdelta283 s_bdelta[6]
#define kdelta293 s_bdelta[7]
#define kdelta307 s_bdelta[8]
#define kdelta311 s_bdelta[9]
#define kdelta313 s_bdelta[10]
#define kdelta317 s_bdelta[11]
#define kdelta331 s_bdelta[12]
#define kdelta337 s_bdelta[13]
#define kdelta347 s_bdelta[14]
#define kdelta349 s_bdelta[15]
#define kdelta353 s_bdelta[16]
#define kdelta359 s_bdelta[17]
#define kdelta367 s_bdelta[18]
#define kdelta373 s_bdelta[19]
#define kdelta379 s_bdelta[20]
#define kdelta383 s_bdelta[21]
#define kdelta389 s_bdelta[22]
#define kdelta397 s_bdelta[23]
#define kdelta401 s_bdelta[24]
#define kdelta409 s_bdelta[25]
#define kdelta419 s_bdelta[26]
#define kdelta421 s_bdelta[27]
#define kdelta431 s_bdelta[28]
#define kdelta433 s_bdelta[29]
#define kdelta439 s_bdelta[30]
#define kdelta443 s_bdelta[31]
#define kdelta449 s_bdelta[32]
#define kdelta457 s_bdelta[33]
#define kdelta461 s_bdelta[34]
#define kdelta463 s_bdelta[35]
#define kdelta467 s_bdelta[36]
#define kdelta479 s_bdelta[37]
#define kdelta487 s_bdelta[38]
#define kdelta491 s_bdelta[39]
#define kdelta499 s_bdelta[40]
#define kdelta503 s_bdelta[41]
#define kdelta509 s_bdelta[42]

  // We let the first 43 threads of each thread block simultaneously transfer
  // kdelta values from global memory to shared memory
  if (threadIdx.x < 43)
    s_bdelta[threadIdx.x] = d_bdelta257[threadIdx.x];

  __syncthreads();

  // One thread, per 256-bit word of bitmap should be launched for this kernel, please.)
  i = blockDim.x * blockIdx.x + threadIdx.x;

  // All threads *must* participate, since they write each other's results to global memory
  {
    unsigned int j;
    unsigned int k;

    // The following handles primes, 256 < p < 512.
    // Since we are executing with one thread per 256-bit word, we will either
    // find one or zero bits to sieve per thread per prime.

    // The bits we sieve will be ORed into one of these eight 32-bit words.
    smap[8*threadIdx.x  ] = 0;
    smap[8*threadIdx.x+1] = 0;
    smap[8*threadIdx.x+2] = 0;
    smap[8*threadIdx.x+3] = 0;
    smap[8*threadIdx.x+4] = 0;
    smap[8*threadIdx.x+5] = 0;
    smap[8*threadIdx.x+6] = 0;
    smap[8*threadIdx.x+7] = 0;

#define SIEVE_256_BIT(p, kdeltap) { \
    j = (i * 256 + p-1 - kdeltap) / p; \
    k = kdeltap + j*p; \
    if ((k>>8) == i) \
      smap[8*threadIdx.x+((k>>5)&7)] |= 1<<(k&31); \
    }

    SIEVE_256_BIT(257, kdelta257);
    SIEVE_256_BIT(263, kdelta263);
    SIEVE_256_BIT(269, kdelta269);
    SIEVE_256_BIT(271, kdelta271);
    SIEVE_256_BIT(277, kdelta277);
    SIEVE_256_BIT(281, kdelta281);
    SIEVE_256_BIT(283, kdelta283);
    SIEVE_256_BIT(293, kdelta293);
    SIEVE_256_BIT(307, kdelta307);
    SIEVE_256_BIT(311, kdelta311);
    SIEVE_256_BIT(313, kdelta313);
    SIEVE_256_BIT(317, kdelta317);
    SIEVE_256_BIT(331, kdelta331);
    SIEVE_256_BIT(337, kdelta337);
    SIEVE_256_BIT(347, kdelta347);
    SIEVE_256_BIT(349, kdelta349);
    SIEVE_256_BIT(353, kdelta353);
    SIEVE_256_BIT(359, kdelta359);
    SIEVE_256_BIT(367, kdelta367);
    SIEVE_256_BIT(373, kdelta373);
    SIEVE_256_BIT(379, kdelta379);
    SIEVE_256_BIT(383, kdelta383);
    SIEVE_256_BIT(389, kdelta389);
    SIEVE_256_BIT(397, kdelta397);
    SIEVE_256_BIT(401, kdelta401);
    SIEVE_256_BIT(409, kdelta409);
    SIEVE_256_BIT(419, kdelta419);
    SIEVE_256_BIT(421, kdelta421);
    SIEVE_256_BIT(431, kdelta431);
    SIEVE_256_BIT(433, kdelta433);
    SIEVE_256_BIT(439, kdelta439);
    SIEVE_256_BIT(443, kdelta443);
    SIEVE_256_BIT(449, kdelta449);
    SIEVE_256_BIT(457, kdelta457);
    SIEVE_256_BIT(461, kdelta461);
    SIEVE_256_BIT(463, kdelta463);
    SIEVE_256_BIT(467, kdelta467);
    SIEVE_256_BIT(479, kdelta479);
    SIEVE_256_BIT(487, kdelta487);
    SIEVE_256_BIT(491, kdelta491);
    SIEVE_256_BIT(499, kdelta499);
    SIEVE_256_BIT(503, kdelta503);
    SIEVE_256_BIT(509, kdelta509);

// Which strategy to write results to global memory?  atomicAnd isn't really necessary, as long
// as all kernel's prior to this one have finished before out kernel runs.
// Scrambling the copies from shared memory to global memory yields ~100% global load/store efficiency,
// but doesn't actually have much performance gain.
// Straightforward copy reports 25% global load/store efficiency, and is only a little slower.

    __syncthreads();    // Make sure everybody has stored their results
    d_bitmapw[8*i-8*threadIdx.x+             threadIdx.x] &= ~smap[             threadIdx.x];
    d_bitmapw[8*i-8*threadIdx.x+  blockDim.x+threadIdx.x] &= ~smap[  blockDim.x+threadIdx.x];
    d_bitmapw[8*i-8*threadIdx.x+2*blockDim.x+threadIdx.x] &= ~smap[2*blockDim.x+threadIdx.x];
    d_bitmapw[8*i-8*threadIdx.x+3*blockDim.x+threadIdx.x] &= ~smap[3*blockDim.x+threadIdx.x];
    d_bitmapw[8*i-8*threadIdx.x+4*blockDim.x+threadIdx.x] &= ~smap[4*blockDim.x+threadIdx.x];
    d_bitmapw[8*i-8*threadIdx.x+5*blockDim.x+threadIdx.x] &= ~smap[5*blockDim.x+threadIdx.x];
    d_bitmapw[8*i-8*threadIdx.x+6*blockDim.x+threadIdx.x] &= ~smap[6*blockDim.x+threadIdx.x];
    d_bitmapw[8*i-8*threadIdx.x+7*blockDim.x+threadIdx.x] &= ~smap[7*blockDim.x+threadIdx.x];
  }
}


__global__ void __launch_bounds__(128, 4) rcv_sieve_small_521_1021(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bdelta521,  /* pointer to list of 75 deltas for primes 521 through 1021 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        )
{
  unsigned int i;
  __shared__ unsigned int smap[16*128]; // Sixteen 32-bit words per thread per block
  __shared__ unsigned int s_bdelta[75]; // Space for 75 k-values in shared (fast) memory
#define kdelta521 s_bdelta[0]
#define kdelta523 s_bdelta[1]
#define kdelta541 s_bdelta[2]
#define kdelta547 s_bdelta[3]
#define kdelta557 s_bdelta[4]
#define kdelta563 s_bdelta[5]
#define kdelta569 s_bdelta[6]
#define kdelta571 s_bdelta[7]
#define kdelta577 s_bdelta[8]
#define kdelta587 s_bdelta[9]
#define kdelta593 s_bdelta[10]
#define kdelta599 s_bdelta[11]
#define kdelta601 s_bdelta[12]
#define kdelta607 s_bdelta[13]
#define kdelta613 s_bdelta[14]
#define kdelta617 s_bdelta[15]
#define kdelta619 s_bdelta[16]
#define kdelta631 s_bdelta[17]
#define kdelta641 s_bdelta[18]
#define kdelta643 s_bdelta[19]
#define kdelta647 s_bdelta[20]
#define kdelta653 s_bdelta[21]
#define kdelta659 s_bdelta[22]
#define kdelta661 s_bdelta[23]
#define kdelta673 s_bdelta[24]
#define kdelta677 s_bdelta[25]
#define kdelta683 s_bdelta[26]
#define kdelta691 s_bdelta[27]
#define kdelta701 s_bdelta[28]
#define kdelta709 s_bdelta[29]
#define kdelta719 s_bdelta[30]
#define kdelta727 s_bdelta[31]
#define kdelta733 s_bdelta[32]
#define kdelta739 s_bdelta[33]
#define kdelta743 s_bdelta[34]
#define kdelta751 s_bdelta[35]
#define kdelta757 s_bdelta[36]
#define kdelta761 s_bdelta[37]
#define kdelta769 s_bdelta[38]
#define kdelta773 s_bdelta[39]
#define kdelta787 s_bdelta[40]
#define kdelta797 s_bdelta[41]
#define kdelta809 s_bdelta[42]
#define kdelta811 s_bdelta[43]
#define kdelta821 s_bdelta[44]
#define kdelta823 s_bdelta[45]
#define kdelta827 s_bdelta[46]
#define kdelta829 s_bdelta[47]
#define kdelta839 s_bdelta[48]
#define kdelta853 s_bdelta[49]
#define kdelta857 s_bdelta[50]
#define kdelta859 s_bdelta[51]
#define kdelta863 s_bdelta[52]
#define kdelta877 s_bdelta[53]
#define kdelta881 s_bdelta[54]
#define kdelta883 s_bdelta[55]
#define kdelta887 s_bdelta[56]
#define kdelta907 s_bdelta[57]
#define kdelta911 s_bdelta[58]
#define kdelta919 s_bdelta[59]
#define kdelta929 s_bdelta[60]
#define kdelta937 s_bdelta[61]
#define kdelta941 s_bdelta[62]
#define kdelta947 s_bdelta[63]
#define kdelta953 s_bdelta[64]
#define kdelta967 s_bdelta[65]
#define kdelta971 s_bdelta[66]
#define kdelta977 s_bdelta[67]
#define kdelta983 s_bdelta[68]
#define kdelta991 s_bdelta[69]
#define kdelta997 s_bdelta[70]
#define kdelta1009 s_bdelta[71]
#define kdelta1013 s_bdelta[72]
#define kdelta1019 s_bdelta[73]
#define kdelta1021 s_bdelta[74]

  // CAUTION:  Following code will not work if threadsPerBlock is less than 64

  // Simultaneously transfer maximum number of kdelta values
  if (threadIdx.x < 64)
    s_bdelta[threadIdx.x   ] = d_bdelta521[threadIdx.x   ];
  if (threadIdx.x < 75-64)
    s_bdelta[threadIdx.x+64] = d_bdelta521[threadIdx.x+64];

  __syncthreads();

  // One thread, per 512-bit word of bitmap should be launched for this kernel, please.)
  i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < ((kcount+511)>>9))  // Excess threads don't participate.
  // All threads *must* participate, since they write each other's results to global memory
  {
    unsigned int j;
    unsigned int k;

    // The following handles primes, 512 < p < 1024.
    // Since we are executing with one thread per 512-bit word, we will either
    // find one or zero bits to sieve per thread per prime.

    // The bits we sieve will be ORed into one of these sixteen 32-bit words.
    smap[16*threadIdx.x   ] = 0;
    smap[16*threadIdx.x+ 1] = 0;
    smap[16*threadIdx.x+ 2] = 0;
    smap[16*threadIdx.x+ 3] = 0;
    smap[16*threadIdx.x+ 4] = 0;
    smap[16*threadIdx.x+ 5] = 0;
    smap[16*threadIdx.x+ 6] = 0;
    smap[16*threadIdx.x+ 7] = 0;
    smap[16*threadIdx.x+ 8] = 0;
    smap[16*threadIdx.x+ 9] = 0;
    smap[16*threadIdx.x+10] = 0;
    smap[16*threadIdx.x+11] = 0;
    smap[16*threadIdx.x+12] = 0;
    smap[16*threadIdx.x+13] = 0;
    smap[16*threadIdx.x+14] = 0;
    smap[16*threadIdx.x+15] = 0;

#define SIEVE_512_BIT(p, kdeltap) { \
    j = (i * 512 + p-1 - kdeltap) / p; \
    k = kdeltap + j*p; \
    if ((k>>9) == i) \
      smap[16*threadIdx.x+((k>>5)&15)] |= 1<<(k&31); \
    }

    SIEVE_512_BIT( 521, kdelta521 );
    SIEVE_512_BIT( 523, kdelta523 );
    SIEVE_512_BIT( 541, kdelta541 );
    SIEVE_512_BIT( 547, kdelta547 );
    SIEVE_512_BIT( 557, kdelta557 );
    SIEVE_512_BIT( 563, kdelta563 );
    SIEVE_512_BIT( 569, kdelta569 );
    SIEVE_512_BIT( 571, kdelta571 );
    SIEVE_512_BIT( 577, kdelta577 );
    SIEVE_512_BIT( 587, kdelta587 );
    SIEVE_512_BIT( 593, kdelta593 );
    SIEVE_512_BIT( 599, kdelta599 );
    SIEVE_512_BIT( 601, kdelta601 );
    SIEVE_512_BIT( 607, kdelta607 );
    SIEVE_512_BIT( 613, kdelta613 );
    SIEVE_512_BIT( 617, kdelta617 );
    SIEVE_512_BIT( 619, kdelta619 );
    SIEVE_512_BIT( 631, kdelta631 );
    SIEVE_512_BIT( 641, kdelta641 );
    SIEVE_512_BIT( 643, kdelta643 );
    SIEVE_512_BIT( 647, kdelta647 );
    SIEVE_512_BIT( 653, kdelta653 );
    SIEVE_512_BIT( 659, kdelta659 );
    SIEVE_512_BIT( 661, kdelta661 );
    SIEVE_512_BIT( 673, kdelta673 );
    SIEVE_512_BIT( 677, kdelta677 );
    SIEVE_512_BIT( 683, kdelta683 );
    SIEVE_512_BIT( 691, kdelta691 );
    SIEVE_512_BIT( 701, kdelta701 );
    SIEVE_512_BIT( 709, kdelta709 );
    SIEVE_512_BIT( 719, kdelta719 );
    SIEVE_512_BIT( 727, kdelta727 );
    SIEVE_512_BIT( 733, kdelta733 );
    SIEVE_512_BIT( 739, kdelta739 );
    SIEVE_512_BIT( 743, kdelta743 );
    SIEVE_512_BIT( 751, kdelta751 );
    SIEVE_512_BIT( 757, kdelta757 );
    SIEVE_512_BIT( 761, kdelta761 );
    SIEVE_512_BIT( 769, kdelta769 );
    SIEVE_512_BIT( 773, kdelta773 );
    SIEVE_512_BIT( 787, kdelta787 );
    SIEVE_512_BIT( 797, kdelta797 );
    SIEVE_512_BIT( 809, kdelta809 );
    SIEVE_512_BIT( 811, kdelta811 );
    SIEVE_512_BIT( 821, kdelta821 );
    SIEVE_512_BIT( 823, kdelta823 );
    SIEVE_512_BIT( 827, kdelta827 );
    SIEVE_512_BIT( 829, kdelta829 );
    SIEVE_512_BIT( 839, kdelta839 );
    SIEVE_512_BIT( 853, kdelta853 );
    SIEVE_512_BIT( 857, kdelta857 );
    SIEVE_512_BIT( 859, kdelta859 );
    SIEVE_512_BIT( 863, kdelta863 );
    SIEVE_512_BIT( 877, kdelta877 );
    SIEVE_512_BIT( 881, kdelta881 );
    SIEVE_512_BIT( 883, kdelta883 );
    SIEVE_512_BIT( 887, kdelta887 );
    SIEVE_512_BIT( 907, kdelta907 );
    SIEVE_512_BIT( 911, kdelta911 );
    SIEVE_512_BIT( 919, kdelta919 );
    SIEVE_512_BIT( 929, kdelta929 );
    SIEVE_512_BIT( 937, kdelta937 );
    SIEVE_512_BIT( 941, kdelta941 );
    SIEVE_512_BIT( 947, kdelta947 );
    SIEVE_512_BIT( 953, kdelta953 );
    SIEVE_512_BIT( 967, kdelta967 );
    SIEVE_512_BIT( 971, kdelta971 );
    SIEVE_512_BIT( 977, kdelta977 );
    SIEVE_512_BIT( 983, kdelta983 );
    SIEVE_512_BIT( 991, kdelta991 );
    SIEVE_512_BIT( 997, kdelta997 );
    SIEVE_512_BIT(1009, kdelta1009);
    SIEVE_512_BIT(1013, kdelta1013);
    SIEVE_512_BIT(1019, kdelta1019);
    SIEVE_512_BIT(1021, kdelta1021);

// Which strategy to write results to global memory?  atomicAnd isn't really necessary, as long
// as all kernel's prior to this one have finished before our kernel runs.
// Scrambling the copies from shared memory to global memory yields ~100% global load/store efficiency,
// but doesn't actually have a huge performance gain.
// Straightforward copy reports 6.25% global load/store efficiency.

    __syncthreads();    // Make sure everybody has stored their results
    d_bitmapw[16*i-16*threadIdx.x+              threadIdx.x] &= ~smap[              threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+   blockDim.x+threadIdx.x] &= ~smap[   blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+ 2*blockDim.x+threadIdx.x] &= ~smap[ 2*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+ 3*blockDim.x+threadIdx.x] &= ~smap[ 3*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+ 4*blockDim.x+threadIdx.x] &= ~smap[ 4*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+ 5*blockDim.x+threadIdx.x] &= ~smap[ 5*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+ 6*blockDim.x+threadIdx.x] &= ~smap[ 6*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+ 7*blockDim.x+threadIdx.x] &= ~smap[ 7*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+ 8*blockDim.x+threadIdx.x] &= ~smap[ 8*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+ 9*blockDim.x+threadIdx.x] &= ~smap[ 9*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+10*blockDim.x+threadIdx.x] &= ~smap[10*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+11*blockDim.x+threadIdx.x] &= ~smap[11*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+12*blockDim.x+threadIdx.x] &= ~smap[12*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+13*blockDim.x+threadIdx.x] &= ~smap[13*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+14*blockDim.x+threadIdx.x] &= ~smap[14*blockDim.x+threadIdx.x];
    d_bitmapw[16*i-16*threadIdx.x+15*blockDim.x+threadIdx.x] &= ~smap[15*blockDim.x+threadIdx.x];
    return;
  }
}


// To limit our shared memory footprint, we launch with a small threads per block.
// Note that we tested with 32 TPB, 64 TPB, and 256 TPB.  In each case, the
// run-time of the kernel remained (at approximately 100 us), and the occupancy
// remained near 1/6.  Should investigate if something is limiting our occupancy.
__global__ void __launch_bounds__(32, 4) rcv_sieve_small_1031_2039(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_kdelta1031,  /* pointer to list of 137 deltas for primes 1031 through 2039 */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        )
{
  unsigned int i;
  __shared__ unsigned int smap[32*32];   // Thirty-two 32-bit words per thread per block
  __shared__ unsigned int s_kdelta[137]; // Space for 137 k-values in shared (fast) memory
#define kdelta1031 s_kdelta[0]
#define kdelta1033 s_kdelta[1]
#define kdelta1039 s_kdelta[2]
#define kdelta1049 s_kdelta[3]
#define kdelta1051 s_kdelta[4]
#define kdelta1061 s_kdelta[5]
#define kdelta1063 s_kdelta[6]
#define kdelta1069 s_kdelta[7]
#define kdelta1087 s_kdelta[8]
#define kdelta1091 s_kdelta[9]
#define kdelta1093 s_kdelta[10]
#define kdelta1097 s_kdelta[11]
#define kdelta1103 s_kdelta[12]
#define kdelta1109 s_kdelta[13]
#define kdelta1117 s_kdelta[14]
#define kdelta1123 s_kdelta[15]
#define kdelta1129 s_kdelta[16]
#define kdelta1151 s_kdelta[17]
#define kdelta1153 s_kdelta[18]
#define kdelta1163 s_kdelta[19]
#define kdelta1171 s_kdelta[20]
#define kdelta1181 s_kdelta[21]
#define kdelta1187 s_kdelta[22]
#define kdelta1193 s_kdelta[23]
#define kdelta1201 s_kdelta[24]
#define kdelta1213 s_kdelta[25]
#define kdelta1217 s_kdelta[26]
#define kdelta1223 s_kdelta[27]
#define kdelta1229 s_kdelta[28]
#define kdelta1231 s_kdelta[29]
#define kdelta1237 s_kdelta[30]
#define kdelta1249 s_kdelta[31]
#define kdelta1259 s_kdelta[32]
#define kdelta1277 s_kdelta[33]
#define kdelta1279 s_kdelta[34]
#define kdelta1283 s_kdelta[35]
#define kdelta1289 s_kdelta[36]
#define kdelta1291 s_kdelta[37]
#define kdelta1297 s_kdelta[38]
#define kdelta1301 s_kdelta[39]
#define kdelta1303 s_kdelta[40]
#define kdelta1307 s_kdelta[41]
#define kdelta1319 s_kdelta[42]
#define kdelta1321 s_kdelta[43]
#define kdelta1327 s_kdelta[44]
#define kdelta1361 s_kdelta[45]
#define kdelta1367 s_kdelta[46]
#define kdelta1373 s_kdelta[47]
#define kdelta1381 s_kdelta[48]
#define kdelta1399 s_kdelta[49]
#define kdelta1409 s_kdelta[50]
#define kdelta1423 s_kdelta[51]
#define kdelta1427 s_kdelta[52]
#define kdelta1429 s_kdelta[53]
#define kdelta1433 s_kdelta[54]
#define kdelta1439 s_kdelta[55]
#define kdelta1447 s_kdelta[56]
#define kdelta1451 s_kdelta[57]
#define kdelta1453 s_kdelta[58]
#define kdelta1459 s_kdelta[59]
#define kdelta1471 s_kdelta[60]
#define kdelta1481 s_kdelta[61]
#define kdelta1483 s_kdelta[62]
#define kdelta1487 s_kdelta[63]
#define kdelta1489 s_kdelta[64]
#define kdelta1493 s_kdelta[65]
#define kdelta1499 s_kdelta[66]
#define kdelta1511 s_kdelta[67]
#define kdelta1523 s_kdelta[68]
#define kdelta1531 s_kdelta[69]
#define kdelta1543 s_kdelta[70]
#define kdelta1549 s_kdelta[71]
#define kdelta1553 s_kdelta[72]
#define kdelta1559 s_kdelta[73]
#define kdelta1567 s_kdelta[74]
#define kdelta1571 s_kdelta[75]
#define kdelta1579 s_kdelta[76]
#define kdelta1583 s_kdelta[77]
#define kdelta1597 s_kdelta[78]
#define kdelta1601 s_kdelta[79]
#define kdelta1607 s_kdelta[80]
#define kdelta1609 s_kdelta[81]
#define kdelta1613 s_kdelta[82]
#define kdelta1619 s_kdelta[83]
#define kdelta1621 s_kdelta[84]
#define kdelta1627 s_kdelta[85]
#define kdelta1637 s_kdelta[86]
#define kdelta1657 s_kdelta[87]
#define kdelta1663 s_kdelta[88]
#define kdelta1667 s_kdelta[89]
#define kdelta1669 s_kdelta[90]
#define kdelta1693 s_kdelta[91]
#define kdelta1697 s_kdelta[92]
#define kdelta1699 s_kdelta[93]
#define kdelta1709 s_kdelta[94]
#define kdelta1721 s_kdelta[95]
#define kdelta1723 s_kdelta[96]
#define kdelta1733 s_kdelta[97]
#define kdelta1741 s_kdelta[98]
#define kdelta1747 s_kdelta[99]
#define kdelta1753 s_kdelta[100]
#define kdelta1759 s_kdelta[101]
#define kdelta1777 s_kdelta[102]
#define kdelta1783 s_kdelta[103]
#define kdelta1787 s_kdelta[104]
#define kdelta1789 s_kdelta[105]
#define kdelta1801 s_kdelta[106]
#define kdelta1811 s_kdelta[107]
#define kdelta1823 s_kdelta[108]
#define kdelta1831 s_kdelta[109]
#define kdelta1847 s_kdelta[110]
#define kdelta1861 s_kdelta[111]
#define kdelta1867 s_kdelta[112]
#define kdelta1871 s_kdelta[113]
#define kdelta1873 s_kdelta[114]
#define kdelta1877 s_kdelta[115]
#define kdelta1879 s_kdelta[116]
#define kdelta1889 s_kdelta[117]
#define kdelta1901 s_kdelta[118]
#define kdelta1907 s_kdelta[119]
#define kdelta1913 s_kdelta[120]
#define kdelta1931 s_kdelta[121]
#define kdelta1933 s_kdelta[122]
#define kdelta1949 s_kdelta[123]
#define kdelta1951 s_kdelta[124]
#define kdelta1973 s_kdelta[125]
#define kdelta1979 s_kdelta[126]
#define kdelta1987 s_kdelta[127]
#define kdelta1993 s_kdelta[128]
#define kdelta1997 s_kdelta[129]
#define kdelta1999 s_kdelta[130]
#define kdelta2003 s_kdelta[131]
#define kdelta2011 s_kdelta[132]
#define kdelta2017 s_kdelta[133]
#define kdelta2027 s_kdelta[134]
#define kdelta2029 s_kdelta[135]
#define kdelta2039 s_kdelta[136]

  // CAUTION:  Following code will not work if threadsPerBlock is less than 32

  // Simultaneously transfer maximum number of kdelta values
  if (threadIdx.x < 32)
  {
    s_kdelta[threadIdx.x    ] = d_kdelta1031[threadIdx.x    ];
    s_kdelta[threadIdx.x+ 32] = d_kdelta1031[threadIdx.x+ 32];
    s_kdelta[threadIdx.x+ 64] = d_kdelta1031[threadIdx.x+ 64];
    s_kdelta[threadIdx.x+ 96] = d_kdelta1031[threadIdx.x+ 96];
  }
  if (threadIdx.x < 137-128)
    s_kdelta[threadIdx.x+128] = d_kdelta1031[threadIdx.x+128];

  __syncthreads();              // Is this necessary?

  // One thread, per 1024-bit word of bitmap should be launched for this kernel, please.)
  i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < ((kcount+1023)>>10))  // Excess threads don't participate.
  // All threads participate.
  {
    unsigned int j;
    unsigned int k;

    // The following handles primes, 1024 < p < 2048.
    // Since we are executing with one thread per 1024-bit word, we will either
    // find one or zero bits to sieve per thread per prime.

    // The bits we sieve will be ORed into one of these thirty-two 32-bit words.
    smap[32*threadIdx.x   ] = 0;
    smap[32*threadIdx.x+ 1] = 0;
    smap[32*threadIdx.x+ 2] = 0;
    smap[32*threadIdx.x+ 3] = 0;
    smap[32*threadIdx.x+ 4] = 0;
    smap[32*threadIdx.x+ 5] = 0;
    smap[32*threadIdx.x+ 6] = 0;
    smap[32*threadIdx.x+ 7] = 0;
    smap[32*threadIdx.x+ 8] = 0;
    smap[32*threadIdx.x+ 9] = 0;
    smap[32*threadIdx.x+10] = 0;
    smap[32*threadIdx.x+11] = 0;
    smap[32*threadIdx.x+12] = 0;
    smap[32*threadIdx.x+13] = 0;
    smap[32*threadIdx.x+14] = 0;
    smap[32*threadIdx.x+15] = 0;
    smap[32*threadIdx.x+16] = 0;
    smap[32*threadIdx.x+17] = 0;
    smap[32*threadIdx.x+18] = 0;
    smap[32*threadIdx.x+19] = 0;
    smap[32*threadIdx.x+20] = 0;
    smap[32*threadIdx.x+21] = 0;
    smap[32*threadIdx.x+22] = 0;
    smap[32*threadIdx.x+23] = 0;
    smap[32*threadIdx.x+24] = 0;
    smap[32*threadIdx.x+25] = 0;
    smap[32*threadIdx.x+26] = 0;
    smap[32*threadIdx.x+27] = 0;
    smap[32*threadIdx.x+28] = 0;
    smap[32*threadIdx.x+29] = 0;
    smap[32*threadIdx.x+30] = 0;
    smap[32*threadIdx.x+31] = 0;

#define SIEVE_1024_BIT(p, kdeltap) { \
    j = (i * 1024 + p-1 - kdeltap) / p; \
    k = kdeltap + j*p; \
    if ((k>>10) == i) \
      smap[32*threadIdx.x+((k>>5)&31)] |= 1<<(k&31); \
    }

#define SIEVE_1024_BIT_DEBUG(p, kdeltap) { \
    j = (i * 1024 + p-1 - kdeltap) / p; \
    k = kdeltap + j*p; \
    if ((k>>10) == i) \
      { \
      smap[32*threadIdx.x+((k>>5)&31)] |= 1<<(k&31); \
      if (k<10000 || k>3140000) \
        printf("p=%u, kdeltap=%u, i=%u, j=%u, k=%u=%8.8X, smap[.]=%8.8X\n", \
                 p, kdeltap, i, j, k, k, \
                 smap[32*threadIdx.x+((k>>5)&31)]); \
      } \
    }

    SIEVE_1024_BIT(1031, kdelta1031);
    SIEVE_1024_BIT(1033, kdelta1033);
    SIEVE_1024_BIT(1039, kdelta1039);
    SIEVE_1024_BIT(1049, kdelta1049);
    SIEVE_1024_BIT(1051, kdelta1051);
    SIEVE_1024_BIT(1061, kdelta1061);
    SIEVE_1024_BIT(1063, kdelta1063);
    SIEVE_1024_BIT(1069, kdelta1069);
    SIEVE_1024_BIT(1087, kdelta1087);
    SIEVE_1024_BIT(1091, kdelta1091);
    SIEVE_1024_BIT(1093, kdelta1093);
    SIEVE_1024_BIT(1097, kdelta1097);
    SIEVE_1024_BIT(1103, kdelta1103);
    SIEVE_1024_BIT(1109, kdelta1109);
    SIEVE_1024_BIT(1117, kdelta1117);
    SIEVE_1024_BIT(1123, kdelta1123);
    SIEVE_1024_BIT(1129, kdelta1129);
    SIEVE_1024_BIT(1151, kdelta1151);
    SIEVE_1024_BIT(1153, kdelta1153);
    SIEVE_1024_BIT(1163, kdelta1163);
    SIEVE_1024_BIT(1171, kdelta1171);
    SIEVE_1024_BIT(1181, kdelta1181);
    SIEVE_1024_BIT(1187, kdelta1187);
    SIEVE_1024_BIT(1193, kdelta1193);
    SIEVE_1024_BIT(1201, kdelta1201);
    SIEVE_1024_BIT(1213, kdelta1213);
    SIEVE_1024_BIT(1217, kdelta1217);
    SIEVE_1024_BIT(1223, kdelta1223);
    SIEVE_1024_BIT(1229, kdelta1229);
    SIEVE_1024_BIT(1231, kdelta1231);
    SIEVE_1024_BIT(1237, kdelta1237);
    SIEVE_1024_BIT(1249, kdelta1249);
    SIEVE_1024_BIT(1259, kdelta1259);
    SIEVE_1024_BIT(1277, kdelta1277);
    SIEVE_1024_BIT(1279, kdelta1279);
    SIEVE_1024_BIT(1283, kdelta1283);
    SIEVE_1024_BIT(1289, kdelta1289);
    SIEVE_1024_BIT(1291, kdelta1291);
    SIEVE_1024_BIT(1297, kdelta1297);
    SIEVE_1024_BIT(1301, kdelta1301);
    SIEVE_1024_BIT(1303, kdelta1303);
    SIEVE_1024_BIT(1307, kdelta1307);
    SIEVE_1024_BIT(1319, kdelta1319);
    SIEVE_1024_BIT(1321, kdelta1321);
    SIEVE_1024_BIT(1327, kdelta1327);
    SIEVE_1024_BIT(1361, kdelta1361);
    SIEVE_1024_BIT(1367, kdelta1367);
    SIEVE_1024_BIT(1373, kdelta1373);
    SIEVE_1024_BIT(1381, kdelta1381);
    SIEVE_1024_BIT(1399, kdelta1399);
    SIEVE_1024_BIT(1409, kdelta1409);
    SIEVE_1024_BIT(1423, kdelta1423);
    SIEVE_1024_BIT(1427, kdelta1427);
    SIEVE_1024_BIT(1429, kdelta1429);
    SIEVE_1024_BIT(1433, kdelta1433);
    SIEVE_1024_BIT(1439, kdelta1439);
    SIEVE_1024_BIT(1447, kdelta1447);
    SIEVE_1024_BIT(1451, kdelta1451);
    SIEVE_1024_BIT(1453, kdelta1453);
    SIEVE_1024_BIT(1459, kdelta1459);
    SIEVE_1024_BIT(1471, kdelta1471);
    SIEVE_1024_BIT(1481, kdelta1481);
    SIEVE_1024_BIT(1483, kdelta1483);
    SIEVE_1024_BIT(1487, kdelta1487);
    SIEVE_1024_BIT(1489, kdelta1489);
    SIEVE_1024_BIT(1493, kdelta1493);
    SIEVE_1024_BIT(1499, kdelta1499);
    SIEVE_1024_BIT(1511, kdelta1511);
    SIEVE_1024_BIT(1523, kdelta1523);
    SIEVE_1024_BIT(1531, kdelta1531);
    SIEVE_1024_BIT(1543, kdelta1543);
    SIEVE_1024_BIT(1549, kdelta1549);
    SIEVE_1024_BIT(1553, kdelta1553);
    SIEVE_1024_BIT(1559, kdelta1559);
    SIEVE_1024_BIT(1567, kdelta1567);
    SIEVE_1024_BIT(1571, kdelta1571);
    SIEVE_1024_BIT(1579, kdelta1579);
    SIEVE_1024_BIT(1583, kdelta1583);
    SIEVE_1024_BIT(1597, kdelta1597);
    SIEVE_1024_BIT(1601, kdelta1601);
    SIEVE_1024_BIT(1607, kdelta1607);
    SIEVE_1024_BIT(1609, kdelta1609);
    SIEVE_1024_BIT(1613, kdelta1613);
    SIEVE_1024_BIT(1619, kdelta1619);
    SIEVE_1024_BIT(1621, kdelta1621);
    SIEVE_1024_BIT(1627, kdelta1627);
    SIEVE_1024_BIT(1637, kdelta1637);
    SIEVE_1024_BIT(1657, kdelta1657);
    SIEVE_1024_BIT(1663, kdelta1663);
    SIEVE_1024_BIT(1667, kdelta1667);
    SIEVE_1024_BIT(1669, kdelta1669);
    SIEVE_1024_BIT(1693, kdelta1693);
    SIEVE_1024_BIT(1697, kdelta1697);
    SIEVE_1024_BIT(1699, kdelta1699);
    SIEVE_1024_BIT(1709, kdelta1709);
    SIEVE_1024_BIT(1721, kdelta1721);
    SIEVE_1024_BIT(1723, kdelta1723);
    SIEVE_1024_BIT(1733, kdelta1733);
    SIEVE_1024_BIT(1741, kdelta1741);
    SIEVE_1024_BIT(1747, kdelta1747);
    SIEVE_1024_BIT(1753, kdelta1753);
    SIEVE_1024_BIT(1759, kdelta1759);
    SIEVE_1024_BIT(1777, kdelta1777);
    SIEVE_1024_BIT(1783, kdelta1783);
    SIEVE_1024_BIT(1787, kdelta1787);
    SIEVE_1024_BIT(1789, kdelta1789);
    SIEVE_1024_BIT(1801, kdelta1801);
    SIEVE_1024_BIT(1811, kdelta1811);
    SIEVE_1024_BIT(1823, kdelta1823);
    SIEVE_1024_BIT(1831, kdelta1831);
    SIEVE_1024_BIT(1847, kdelta1847);
    SIEVE_1024_BIT(1861, kdelta1861);
    SIEVE_1024_BIT(1867, kdelta1867);
    SIEVE_1024_BIT(1871, kdelta1871);
    SIEVE_1024_BIT(1873, kdelta1873);
    SIEVE_1024_BIT(1877, kdelta1877);
    SIEVE_1024_BIT(1879, kdelta1879);
    SIEVE_1024_BIT(1889, kdelta1889);
    SIEVE_1024_BIT(1901, kdelta1901);
    SIEVE_1024_BIT(1907, kdelta1907);
    SIEVE_1024_BIT(1913, kdelta1913);
    SIEVE_1024_BIT(1931, kdelta1931);
    SIEVE_1024_BIT(1933, kdelta1933);
    SIEVE_1024_BIT(1949, kdelta1949);
    SIEVE_1024_BIT(1951, kdelta1951);
    SIEVE_1024_BIT(1973, kdelta1973);
    SIEVE_1024_BIT(1979, kdelta1979);
    SIEVE_1024_BIT(1987, kdelta1987);
    SIEVE_1024_BIT(1993, kdelta1993);
    SIEVE_1024_BIT(1997, kdelta1997);
    SIEVE_1024_BIT(1999, kdelta1999);
    SIEVE_1024_BIT(2003, kdelta2003);
    SIEVE_1024_BIT(2011, kdelta2011);
    SIEVE_1024_BIT(2017, kdelta2017);
    SIEVE_1024_BIT(2027, kdelta2027);
    SIEVE_1024_BIT(2029, kdelta2029);
    SIEVE_1024_BIT(2039, kdelta2039);

// Which strategy to write results to global memory?  atomicAnd isn't really necessary, as long
// as all kernel's prior to this one have finished before our kernel runs.
// Scrambling the copies from shared memory to global memory yields ~100% global load/store efficiency,
// but doesn't actually have a huge performance gain.
// Straightforward copy reports 6.25% global load/store efficiency.

#if 0                                            // NVVP reported 211.773 us, 4.6%/12.5% global load/store efficiency
    d_bitmapw[32*i   ] &= ~smap[32*threadIdx.x   ];
    d_bitmapw[32*i+ 1] &= ~smap[32*threadIdx.x+ 1];
    d_bitmapw[32*i+ 2] &= ~smap[32*threadIdx.x+ 2];
    d_bitmapw[32*i+ 3] &= ~smap[32*threadIdx.x+ 3];
    d_bitmapw[32*i+ 4] &= ~smap[32*threadIdx.x+ 4];
    d_bitmapw[32*i+ 5] &= ~smap[32*threadIdx.x+ 5];
    d_bitmapw[32*i+ 6] &= ~smap[32*threadIdx.x+ 6];
    d_bitmapw[32*i+ 7] &= ~smap[32*threadIdx.x+ 7];
    d_bitmapw[32*i+ 8] &= ~smap[32*threadIdx.x+ 8];
    d_bitmapw[32*i+ 9] &= ~smap[32*threadIdx.x+ 9];
    d_bitmapw[32*i+10] &= ~smap[32*threadIdx.x+10];
    d_bitmapw[32*i+11] &= ~smap[32*threadIdx.x+11];
    d_bitmapw[32*i+12] &= ~smap[32*threadIdx.x+12];
    d_bitmapw[32*i+13] &= ~smap[32*threadIdx.x+13];
    d_bitmapw[32*i+14] &= ~smap[32*threadIdx.x+14];
    d_bitmapw[32*i+15] &= ~smap[32*threadIdx.x+15];
    d_bitmapw[32*i+16] &= ~smap[32*threadIdx.x+16];
    d_bitmapw[32*i+17] &= ~smap[32*threadIdx.x+17];
    d_bitmapw[32*i+18] &= ~smap[32*threadIdx.x+18];
    d_bitmapw[32*i+19] &= ~smap[32*threadIdx.x+19];
    d_bitmapw[32*i+20] &= ~smap[32*threadIdx.x+20];
    d_bitmapw[32*i+21] &= ~smap[32*threadIdx.x+21];
    d_bitmapw[32*i+22] &= ~smap[32*threadIdx.x+22];
    d_bitmapw[32*i+23] &= ~smap[32*threadIdx.x+23];
    d_bitmapw[32*i+24] &= ~smap[32*threadIdx.x+24];
    d_bitmapw[32*i+25] &= ~smap[32*threadIdx.x+25];
    d_bitmapw[32*i+26] &= ~smap[32*threadIdx.x+26];
    d_bitmapw[32*i+27] &= ~smap[32*threadIdx.x+27];
    d_bitmapw[32*i+28] &= ~smap[32*threadIdx.x+28];
    d_bitmapw[32*i+29] &= ~smap[32*threadIdx.x+29];
    d_bitmapw[32*i+30] &= ~smap[32*threadIdx.x+30];
    d_bitmapw[32*i+31] &= ~smap[32*threadIdx.x+31];
    return;
#else                                            // NVVP reported        us,    % global load/store efficiency
    __syncthreads();    // Make sure everybody has stored their results
    d_bitmapw[32*i-32*threadIdx.x+              threadIdx.x] &= ~smap[              threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+   blockDim.x+threadIdx.x] &= ~smap[   blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+ 2*blockDim.x+threadIdx.x] &= ~smap[ 2*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+ 3*blockDim.x+threadIdx.x] &= ~smap[ 3*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+ 4*blockDim.x+threadIdx.x] &= ~smap[ 4*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+ 5*blockDim.x+threadIdx.x] &= ~smap[ 5*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+ 6*blockDim.x+threadIdx.x] &= ~smap[ 6*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+ 7*blockDim.x+threadIdx.x] &= ~smap[ 7*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+ 8*blockDim.x+threadIdx.x] &= ~smap[ 8*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+ 9*blockDim.x+threadIdx.x] &= ~smap[ 9*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+10*blockDim.x+threadIdx.x] &= ~smap[10*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+11*blockDim.x+threadIdx.x] &= ~smap[11*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+12*blockDim.x+threadIdx.x] &= ~smap[12*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+13*blockDim.x+threadIdx.x] &= ~smap[13*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+14*blockDim.x+threadIdx.x] &= ~smap[14*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+15*blockDim.x+threadIdx.x] &= ~smap[15*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+16*blockDim.x+threadIdx.x] &= ~smap[16*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+17*blockDim.x+threadIdx.x] &= ~smap[17*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+18*blockDim.x+threadIdx.x] &= ~smap[18*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+19*blockDim.x+threadIdx.x] &= ~smap[19*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+20*blockDim.x+threadIdx.x] &= ~smap[20*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+21*blockDim.x+threadIdx.x] &= ~smap[21*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+22*blockDim.x+threadIdx.x] &= ~smap[22*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+23*blockDim.x+threadIdx.x] &= ~smap[23*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+24*blockDim.x+threadIdx.x] &= ~smap[24*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+25*blockDim.x+threadIdx.x] &= ~smap[25*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+26*blockDim.x+threadIdx.x] &= ~smap[26*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+27*blockDim.x+threadIdx.x] &= ~smap[27*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+28*blockDim.x+threadIdx.x] &= ~smap[28*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+29*blockDim.x+threadIdx.x] &= ~smap[29*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+30*blockDim.x+threadIdx.x] &= ~smap[30*blockDim.x+threadIdx.x];
    d_bitmapw[32*i-32*threadIdx.x+31*blockDim.x+threadIdx.x] &= ~smap[31*blockDim.x+threadIdx.x];
    return;
#endif

  }
}


__global__ void __launch_bounds__(256, 4) rcv_sieve_primes(
        unsigned int tidoffseta,/* Offset from tid to first tid's element in tree */
        unsigned int tidoffsetz,/* Offset from tid to just past last tid's element in tree */
        unsigned int *d_plist,  /* pointer to list of primes */
        unsigned int pcount,    /* number of elements in list */
        int96        kstart,    /* lowest k-value in upcoming sieve */
        unsigned int kcount,    /* number of bits in upcoming sieve */
        unsigned int *d_bdelta, /* pointer to starting k-values per prime */
        unsigned int *d_kncount,/* pointer to count of k-values per prime */
        unsigned int *d_ktree,  /* pointer to tree of count of k-values per prime */
        unsigned int *d_bitmapw /* bitmap for the sieve, 32-bit words */
        )
{
  unsigned int i;
  unsigned int j;
  unsigned int k;
  unsigned int l;

  unsigned int pcountpow2;      /* next power of 2 >= pcount */

  for (pcountpow2=1; pcountpow2 < pcount; pcountpow2 = pcountpow2+pcountpow2)
    ;

  unsigned int ndeeper;         /* number of elements 1 level deeper in tree */
  ndeeper = pcount+pcount - pcountpow2;

  i = blockDim.x * blockIdx.x + threadIdx.x;
  i += tidoffseta;              /* Work the tree starting at this offset */

  if ((i < d_ktree[1]) && (i < tidoffsetz))  // Root of tree contains total threads.
  {
    j = 1;      // Root of tree.  Contains no useful data
    while (j < pcount)
    {
      j += j;
                // j is even.  Node we seek must be j or j+1 (or descendents)
      if (i >= d_ktree[j])
      {
        i -= d_ktree[j];
        j += 1;
      }
      //if (i >= d_ktree[j])
      //  ***ERROR***   // Cannot occur
    }

    if (j < pcountpow2)
      l = j + ndeeper - pcount;
    else
      l = j - pcountpow2;


                        // l now indexes the prime we are sieving
                        // i contains the instance of sieving with this prime

    k = d_bdelta[l] + i*d_plist[l];  // Get the bit (relative to the entire sieve) we are to clear


    // Following two methods both work.  For repeatability, we prefer atomicAnd.
    // Not entirely certain about undocumented CUDA behavior, but we don't
    // believe atomicAnd slows us down.  CUDA generally claims stalls only
    // occur when you use the result of an instruction.  We are careful to
    // *avoid* use of the atomicAnd result.  Only stall should occur if another
    // thread coincidentally needs to reference the same word before hardware
    // (memory controller?) completes the operation.
    // BTW, all CUDA since Compute Capability 1.1 support atomicAnd.
#if 1
    if (k < kcount)
      if (d_bitmapw[k>>5] & 0x00000001<<(k&31)) // Is our bit still on?
        atomicAnd(&d_bitmapw[k>>5], ~(0x00000001<<(k&31))); // If yes, turn it off.  If no, save atomic op

                        // Note:  Avoid unlocked word-size AND operation.
                        // The probability of a collission is 4 times as
                        // large, compared with byte-size AND operation.
                        // So, we'll run more excess divisibility tests.
#else    // This section uses byte-wide bitmap
    if (k < kcount)
      if (((unsigned char *)d_bitmapw)[k>>3] & 0x01<<(k&7)) // Is our bit still on?
        ((unsigned char *)d_bitmapw)[k>>3] &= ~(0x01<<(k&7));  // If yes, turn it off.  If no, save a memory write.

                        // Note:  The above is not locked.  Other
                        // concurrent threads may be changing the same byte.
                        // If we lose an occasional clear, it's no big deal.
                        // We'll just run a few extra divisibility tests.
                        // However, please check any code or resources that
                        // depend on the law of large numbers.
#endif
  }

}



// This kernel simply initializes a small array used as a set of atomic
// indexes into a large array which will collect our linearized candidate.
__global__ void rcv_reset_atomic_indexes(
        unsigned int width,          /* width of atomic index array */
        unsigned int *d_xaindexes)   /* atomic index into array */
{
  unsigned int i;
  
  // One thread, per index should be launched for this kernel, please.)
  // Note:  As implemented in this program, the array is 1 column wide
  i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < width)                /* Excess threads do not participate */
    d_xaindexes[i] = 0;         /* each atomic index is initialized to zero */
}


// This kernel converts a completed set of sieve bits to an array of k-values
// for the trial factor component.
__global__ void __launch_bounds__(256, 4) rcv_linearize_sieve(
        int96        kstart,    /* lowest k-value in current sieve */
        unsigned int kcount,    /* number of bits in current sieve */
        unsigned int *d_bitmapw,/* bitmap for the sieve, 32-bit words */
        unsigned int *d_karray, /* linear array of k-values */
        unsigned int kasize,    /* number of spots in output array */
        unsigned int *d_kaindex /* atomic allocation index into karray */
        )
{
  unsigned int i;
  __shared__ volatile unsigned short bitcount[256];  // Each thread of our block puts bit-counts here
#define LINEARIZE_SMEM_BOUND (32*256*37/100)         // Space for 256 Threads Per Blocks at 37% sieve rate.
                                                     // at MAX_SIEVE_PRIMES =  304, 35.35% survival rate.
                                                     // at MAX_SIEVE_PRIMES =  559, 32.42% survival rate.
                                                     // at MAX_SIEVE_PRIMES = 2500, 26.96% survival rate.
  __shared__ unsigned short smem[LINEARIZE_SMEM_BOUND];
  __shared__ unsigned int   kaix;                    // Index into caller's array for our candidates

  // One thread, per 32-bit word of bitmap should be launched for this kernel, please.)
  i = blockDim.x * blockIdx.x + threadIdx.x;

#if 0
  if (i==0)
    printf("Reached rcv_linearize_sieve\n");
#endif

  if (i < (kcount>>5))	// Excess threads don't participate
  {

    // Count number of bits in use
    {
      unsigned int t;
      t = d_bitmapw[i];
      t = (t&0x55555555) + ((t>> 1)&0x55555555);  // Generate sixteen 2-bit sums
      t = (t&0x33333333) + ((t>> 2)&0x33333333);  // Generate eight 3-bit sums
      t = (t&0x07070707) + ((t>> 4)&0x07070707);  // Generate four 4-bit sums
      t = (t&0x000f000f) + ((t>> 8)&0x000f000f);  // Generate two 5-bit sums
      t = (t&0x0000001f) + ((t>>16)&0x0000001f);  // Generate one 6-bit sum

      bitcount[threadIdx.x] = t;	// Tell everybody how much space my thread needs
    }

    __syncthreads();    // Synchronization required!

    // CAUTION:  Following requires 256 threads per block

    // First five tallies remain within one warp.  Should be in lock-step.
    if (!(i&1))       // If we are running on any thread 0bxxxxxxx1, tally neighbor's count.
      bitcount[threadIdx.x] += bitcount[threadIdx.x + 1];

    if (!(i&2))        // If we are running on any thread 0bxxxxxx1x, tally neighbor's count.
      bitcount[threadIdx.x] += bitcount[threadIdx.x + 2 & ~1];

    if (!(i&4))        // If we are running on any thread 0bxxxxx1xx, tally neighbor's count.
      bitcount[threadIdx.x] += bitcount[threadIdx.x + 4 & ~3];

    if (!(i&8))        // If we are running on any thread 0bxxxx1xxx, tally neighbor's count.
      bitcount[threadIdx.x] += bitcount[threadIdx.x + 8 & ~7];

    if (!(i&16))       // If we are running on any thread 0bxxx1xxxx, tally neighbor's count.
      bitcount[threadIdx.x] += bitcount[threadIdx.x + 16 & ~15];

    // Further tallies are across warps.  Must synchronize
    __syncthreads();   // Synchronization required!
    if (!(i&32))       // If we are running on any thread 0bxx1xxxxx, tally neighbor's count.
      bitcount[threadIdx.x] += bitcount[threadIdx.x + 32 & ~31];

    __syncthreads();   // Synchronization required!
    if (!(i&64))       // If we are running on any thread 0bx1xxxxxx, tally neighbor's count.
      bitcount[threadIdx.x] += bitcount[threadIdx.x + 64 & ~63];

    __syncthreads();   // Synchronization required!
    if (!(i&128))       // If we are running on any thread 0b1xxxxxxx, tally neighbor's count.
      bitcount[threadIdx.x] += bitcount[threadIdx.x + 128 & ~127];

    // At this point, bitcount[...] should contain the total number of bits for the indexed
    // thread plus all high-numbered threads.  I.e., bitcount[0] is the total count.


    // Atomically allocate space in final array for list of k-values
    // One thread allocates space for entire thread block.  Should minimize contention.
    {
      if (threadIdx.x == 0)     // First thread of the thread block?
      {
        kaix = atomicAdd(d_kaindex, bitcount[0]);  // Obtain space in final array for our k-values
        if (kaix + bitcount[0] >= kasize)
          asm("{trap; \n\t}");    // Trap if allocation exceeds array size
        while (kaix + bitcount[0] >= kasize)
          ;    // If trap fails, cause a kernel timeout.
      }
      __syncthreads();
    }

    if (bitcount[0] > LINEARIZE_SMEM_BOUND)    // Will we overshoot our shared memory?
    {
      // This should *rarely* happen -- as when tweaking the code.
      // We revert to a simpler, slower method that doesn't use shared memory
      unsigned int k;
      unsigned int bitmapw;
      unsigned int mykaix;        // karray index for this thread

      k = 32*i;                   // This thread's starting k-value
      bitmapw = d_bitmapw[i];     // 32-bit word containing this thread's bits
      mykaix = kaix + bitcount[0] - bitcount[threadIdx.x];  // Storage index to hold this thread's k-values

      for (int j=0; j<32; j+=1)
      {
        if (bitmapw&1)
          d_karray[mykaix++] = k+j;  // Store current k-value to global memory array
        bitmapw >>= 1;               // Shift sieve bits over
      }

      return;     // No fanfare.  But skip the fast method, which follows
    }


    // Work through one 32-bit word
    {
      unsigned int k;
      unsigned int bitmapw;
      unsigned int six;

      __syncthreads();
      // k = 32*i;                 // This thread's starting k-value
      k = 32*threadIdx.x;          // This thread's starting k-value
      bitmapw = d_bitmapw[i];   // 32-bit word containing this thread's bits
      six = bitcount[0] - bitcount[threadIdx.x];  // Storage index to hold this thread's k-values

      // Unroll this loop, please
      for (int j=0; j<32; j+=1)
      {
        if (bitmapw&1)
          smem[six++] = k+j;   // Store current k-value to shared memory array
        bitmapw >>= 1;         // Shift sieve bits over
      }

    }

    // Here, all warps of our thread block have placed their candidates in shared memory.

    // The role of threads changes to maximize performance in copying
    // those candidates to global memory
    __syncthreads();              // The smem we read probably is not the smem we wrote!

    {
      unsigned int mykaix;        // karray index for this thread
               int six;           // smem index for this thread

      mykaix = (kaix/32)*32 + threadIdx.x;    // Align our threads to karray alignment
                                              // (thread 32n+5 will access word 32m+5)
      six = mykaix - kaix;    // six==0 where mykaix==kaix

      // First set of transfers may not involve lowest thread IDs
      if ((six >= 0) && (six < bitcount[0]))
      {
        d_karray[mykaix] = smem[six]+32*(i-threadIdx.x);    // Copy a candidate
      }

      mykaix += blockDim.x;
      six += blockDim.x;

      // Copy any additional candidates
      for ( ; six < bitcount[0]; six += blockDim.x, mykaix += blockDim.x)
      {
        d_karray[mykaix] = smem[six]+32*(i-threadIdx.x);    // Copy a candidate
      }

    }

  }
}
